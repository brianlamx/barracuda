#include "hip/hip_runtime.h"
/*

   Barracuda - A Short Sequence Aligner for NVIDIA Graphics Cards

   Module: barracuda.cu  - CUDA alignment and samse kernels

   Copyright (C) 2012, University of Cambridge Metabolic Research Labs.
   Contributers: Petr Klus, Dag Lyberg, Simon Lam and Brian Lam

   This program is free software; you can redistribute it and/or
   modify it under the terms of the GNU General Public License
   as published by the Free Software Foundation; either version 3
   of the License, or (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program; if not, write to the Free Software
   Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.

   This program is based on a modified version of BWA
   File Creation date: 2012.6.8

*/

/* (0.7.0) beta: 
  27 Feb 2015 WBL remove bulk loopcount==0 debug
  26 Feb 2015 WBL swap back from bwt_cuda_occ4.cuh to stub bwt_cuda_occ4()
  25 Feb 2015 WBL skip r1.32, r1.33(sequence_shift81 perhaps do later?),
r1.34(sequence_global), r1.35-43, apply r1.43(no nulls in properties.name),
skip r1.45-46, apply r1.47-50(pad bwt to 16 int), 
skip r1.51-55(many_blocks) r1.56(threads_per_sequence) skip r1.57
skip r1.58-59(TotalCores, include hip/hip_runtime_api.h)
apply r1.60-63(d_mycache4, include read_mycache.cuh) skip r1.64
skip r1.65-69(cache_threads, kl_split, kl_par)
  29 Dec 2014 WBL Avoid binary files by removing nulls in properties.name
  21 Feb 2015 WBL reduce volume of debug output
  19 Feb 2015 WBL still no progress... have wound back to r1.89
  try adding huge debug to each kernel launch
  12 Feb 2015 WBL Add displaying timing info for cuda_inexact_match_caller
  fix r1.85 performance problem with include bwt_cuda_occ4.cuh
Split history barracuda_src.cu,v barracuda.cu,v
  11 Feb 2015 WBL Add same_length to copy_sequences_to_cuda_memory
  replaces r1.30 14 Dec 2014 WBL for direct_index force all sequences to be same length
  Add stub bwt_cuda_occ4 add direct_sequence
  move remaining cuda device code for cuda_split_inexact_match_caller etc to cuda2.cuh
  10 Feb 2015 WBL Split history barracuda_src.cu,v barracuda.cu,v
Re-apply r1.25 free kl_host/kl_device, size_t, remove bwtkl_t (now in barracuda.h),
improve "[aln_debug] bwt loaded %lu bytes, <assert.h> include cuda.cuh
  25 Nov 2014 WBL Re-enable cuda_find_exact_matches changes. Note where sequence matches exactly once no longer report other potential matches
  21 Nov 2014 WBL disable cuda_find_exact_matches changes and add <<<>>> logging comments
                  Add header to text .sai file
  19 Nov 2014 WBL merge text and binary output, ie add stdout_aln_head stdout_barracuda_aln1
                  Explicitly clear unused parts of alignment records in binary .sai output file
  13 Nov 2014 WBL try re-enabling cuda_find_exact_matches
  13 Nov 2014 WBL ensure check status of all host cuda calls
  Ensure all kernels followed by hipDeviceSynchronize so they can report asynchronous errors
*/

#define PACKAGE_VERSION "0.7.0 beta $Revision: 1.101 $"
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <stdint.h>
#include <assert.h>
#include "bwtaln.h"
#include "bwtgap.h"
#include "utils.h"
#include "barracuda.h"

#define d_mycache4 const uint4* mycache0
#define d_mycache8 const uint2* mycache0
#define d_mycache16 const uint32_t* mycache0

#define max_mycache 1
#include "read_mycache.cuh"
#undef max_mycache

#undef d_mycache4
#undef d_mycache8
#undef d_mycache16

#define d_mycache4 const uint4* mycache0
#define d_mycache8 const uint2* mycache0
#define d_mycache16 const uint32_t* mycache0,const uint32_t* mycache1

#define max_mycache 2
#include "read_mycache.cuh"
#undef max_mycache

#undef d_mycache4
#undef d_mycache8
#undef d_mycache16

#define d_mycache4 const uint4* mycache0
#define d_mycache8 const uint2* mycache0,const uint2* mycache1
#define d_mycache16 const uint32_t* mycache0,const uint32_t* mycache1,const uint32_t* mycache2,const uint32_t* mycache3

#define max_mycache 4
#include "read_mycache.cuh"
#undef max_mycache

#undef d_mycache4
#undef d_mycache8
#undef d_mycache16

#define d_mycache4 const uint4* mycache0,const uint4* mycache1
#define d_mycache8 const uint2* mycache0,const uint2* mycache1,const uint2* mycache2,const uint2* mycache3
#define d_mycache16 const uint32_t* mycache0,const uint32_t* mycache1,const uint32_t* mycache2,const uint32_t* mycache3,const uint32_t* mycache4,const uint32_t* mycache5,const uint32_t* mycache6,const uint32_t* mycache7

#define max_mycache 8
#include "read_mycache.cuh"
#undef max_mycache

#undef d_mycache4
#undef d_mycache8
#undef d_mycache16

#define d_mycache4 const uint4* mycache0,const uint4* mycache1,const uint4* mycache2,const uint4* mycache3
#define d_mycache8 const uint2* mycache0,const uint2* mycache1,const uint2* mycache2,const uint2* mycache3,const uint2* mycache4,const uint2* mycache5,const uint2* mycache6,const uint2* mycache7
#define d_mycache16 const uint32_t* mycache0,const uint32_t* mycache1,const uint32_t* mycache2,const uint32_t* mycache3,const uint32_t* mycache4,const uint32_t* mycache5,const uint32_t* mycache6,const uint32_t* mycache7,const uint32_t* mycache8,const uint32_t* mycache9,const uint32_t* mycache10,const uint32_t* mycache11,const uint32_t* mycache12,const uint32_t* mycache13,const uint32_t* mycache14,const uint32_t* mycache15

#define max_mycache 16
#include "read_mycache.cuh"
#undef max_mycache

#undef d_mycache4
#undef d_mycache8
#undef d_mycache16

#include "barracuda.cuh"

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////

// Performance switches
#define BWT_2_OCC_ENABLE 0 // enable looking up of k and l in the same time for counting character occurrence (slower, so disable by default)
#define BWT_TABLE_LOOKUP_ENABLE 1 // use lookup table when instead of counting character occurrence, (faster so enable by default)

//The followings are settings for memory allocations and memory requirements
#define MIN_MEM_REQUIREMENT 768 // minimal global memory requirement in (MiB).  Currently at 768MB
#define CUDA_TESLA 1350 // enlarged workspace buffer. Currently at 1350MB will be halved if not enough mem available

#define SEQUENCE_TABLE_SIZE_EXPONENTIAL 23// DO NOT CHANGE! buffer size in (2^)units for sequences and alignment storages (batch size)
// Maximum exponential is up to 30 [~ 1  GBytes] for non-debug, non alignment
// Maximum exponential is up to 26 [~ 128MBytes] for debug
// Maximum exponential is up to 23 for alignment with 4GB RAM(default : 23)

//The followings are for DEBUG only
#define CUDA_SAMSE 0 //Enable CUDA SAMSE code, debug only (leave ON)

// how much debugging information shall the kernel output? kernel output only works for fermi and above
#define DEBUG_LEVEL 0
#define USE_PETR_SPLIT_KERNEL 0
// how long should a subsequence be for one kernel launch


//Global variables for inexact match <<do not change>>
#define STATE_M 0
#define STATE_I 1
#define STATE_D 2


//macro was beyound comprehension
inline void write_to_half_byte_array(unsigned char * array, const int index, const int data) {
  const int wordindex = index>>3;
  const int byteindex = wordindex*4 + ((index>>1) & 0x3);
  if((index)&0x1) array[byteindex] = (array[byteindex]&0xF0) | (data &0x0F);
  else            array[byteindex] = (array[byteindex]&0x0F) | (data<<4);
}

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////

//CUDA global variables
__device__ __constant__ bwt_t bwt_cuda;
__device__ __constant__ bwt_t rbwt_cuda;
__device__ __constant__ uint32_t* bwt_occ_array2;
__device__ __constant__ gap_opt_t options_cuda;

//Texture Maps
// uint4 is used because the maximum width for CUDA texture bind of 1D memory is 2^27,
// and uint4 the structure 4xinteger is x,y,z,w coordinates and is 16 bytes long,
// therefore effectively there are 2^27x16bytes memory can be access = 2GBytes memory.
texture<uint4, 1, hipReadModeElementType> bwt_occ_array;
texture<uint4, 1, hipReadModeElementType> rbwt_occ_array;
texture<unsigned int, 1, hipReadModeElementType> sequences_array;
texture<uint2, 1, hipReadModeElementType> sequences_index_array;

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////

void report_cuda_error_GPU(const char *message)
{
	hipError_t cuda_err = hipGetLastError();

	if(hipSuccess != cuda_err)
	{
		fprintf(stderr,"%s\n",message);
		fprintf(stderr,"%s\n", hipGetErrorString(cuda_err));
		exit(1);
	}
}

void report_cuda_error_GPU(hipError_t cuda_error, const char *message)
{
	if(hipSuccess != cuda_error)
	{
		fprintf(stderr,"%s\n",message);
		fprintf(stderr,"%s\n", hipGetErrorString(cuda_error));
		exit(1);
	}
}

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////


size_t copy_bwts_to_cuda_memory(const char * prefix, uint32_t ** bwt, uint32_t mem_available, bwtint_t* seq_len)
// bwt occurrence array to global and bind to texture, bwt structure to constant memory
// this function only load part of the bwt for alignment only.  SA is not loaded.
// mem available in MiB (not bytes)
{
	bwt_t * bwt_src;
	size_t size_read = 0;

	#if DEBUG_LEVEL > 0
			fprintf(stderr,"[aln_debug] mem left: %d\n", mem_available);
	#endif

	//Original BWT
	//Load bwt occurrence array from from disk
	char *str = (char*)calloc(strlen(prefix) + 10, 1);
	strcpy(str, prefix); strcat(str, ".bwt");
	bwt_src = bwt_restore_bwt(str);
	free(str);

	#if DEBUG_LEVEL > 0
			fprintf(stderr,"[aln_debug] bwt loaded %lu bytes to CPU \n", size_read);
	#endif
	size_read = bwt_src->bwt_size*sizeof(uint32_t);
	mem_available = mem_available - uint32_t (size_read>>20); // mem available in MiB (not bytes)
	*seq_len = bwt_src->seq_len;

	if(mem_available > 0)
	{
		//Allocate memory for bwt
		const int bwt_size = (bwt_src->bwt_size + 15) & (~0xf); //ensure multiple of 16 ints
		//printf("bwt_size %d padded to %d uint32_t for FIXED_MAX_global_bwt\n",bwt_src->bwt_size,bwt_size);
		hipMalloc((void**)bwt, bwt_size*sizeof(uint32_t));
		report_cuda_error_GPU("[aln_core] Error allocating memory for \"bwt_occurrence array\".\n");
		hipMemset(&((*bwt)[bwt_size-16]), 0, 16*sizeof(uint32_t));
		report_cuda_error_GPU("[aln_core] Error clearing padding in \"bwt_occurrence array\".\n");
		//copy bwt occurrence array from host to device and dump the bwt to save CPU memory
		hipMemcpy (*bwt, bwt_src->bwt, bwt_src->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error copying  \"bwt occurrence array\" to GPU.\n");

		#if DEBUG_LEVEL > 0
			fprintf(stderr,"[aln_debug] bwt loaded to GPU \n");
			fprintf(stderr,"[aln_debug] bwtsize in MiB %u\n",(bwt_src->bwt_size*sizeof(uint32_t)) >>20);
		#endif

		//copy bwt structure data to constant memory bwt_cuda structure
		hipMemcpyToSymbol(HIP_SYMBOL( bwt_cuda), bwt_src, sizeof(bwt_t), 0, hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error binding  \"bwt_src\" to bwt_cuda constant.\n");

		//free bwt_src from memory
		bwt_destroy(bwt_src);


	}
	else
	{
		fprintf(stderr,"[aln_core] Not enough device memory to perform alignment.\n");
		//free bwt_src from memory
		bwt_destroy(bwt_src);
		return 0;
	}

	#if DEBUG_LEVEL > 0
			fprintf(stderr,"[aln_debug] bwt loaded, mem left: %d MiB\n", mem_available);
	#endif

	return size_read;
}

void free_bwts_from_cuda_memory( unsigned int * bwt)
{
	if ( bwt != 0 )
	{
		//hipUnbindTexture(bwt_occ_array);
		hipFree(bwt);
	}
}

void swap(bwt_aln1_t *x, bwt_aln1_t *y)
{
   bwt_aln1_t temp;
   temp = *x;
   *x = *y;
   *y = temp;
}

int choose_pivot(int i,int j)
{
   return((i+j) /2);
}

void aln_quicksort(bwt_aln1_t *aln, int m, int n)
//This function sorts the alignment array from barracuda to make it compatible with SAMSE/SAMPE cores
{
	int key,i,j,k;

	if (m < n)
	{
	      k = choose_pivot(m, n);
	      swap(&aln[m],&aln[k]);
	      key = aln[m].score;
	      i = m+1;
	      j = n;
	      while(i <= j)
	      {
	         while((i <= n) && (aln[i].score <= key))
	                i++;
	         while((j >= m) && (aln[j].score > key))
	                j--;
	         if(i < j)
	                swap(&aln[i],&aln[j]);
	      }
	      // swap two elements
	      swap(&aln[m],&aln[j]);
	      // recursively sort the lesser lists
	      aln_quicksort(aln, m, j-1);
	      aln_quicksort(aln, j+1, n);
	 }
}

///new sorting and sequence input code


//TODO starts here!!!!!!!!!!!

void barracuda_sort_queries(bwa_seq_t *seqs, unsigned int *order)
{
	return;
}

inline void barracuda_seq_reverse(int len, char *seq)
{
	int i;
	for (i = 0; i < len>>1; ++i) {
			char tmp = seq[len-1-i];
			seq[len-1-i] = seq[i]; seq[i] = tmp;
	}
}

inline void  barracuda_write_to_half_byte_array(barracuda_query_array_t *seqs, unsigned char *half_byte_array, uint2 *main_sequences_index, int nseqs)
{
	int i, j = 0;
	int start_pos = 0;

	for (i = 0; i < nseqs; i++){
		barracuda_query_array_t * p = seqs + i;
		//moved from barracuda_read_seq.  Reason: sorting needed to be done from the back.
		barracuda_seq_reverse(p->len, p->seq); // *IMPORTANT*: will be reversed back in bwa_refine_gapped()
		main_sequences_index[i].x = start_pos;
		main_sequences_index[i].y = p->len;
		for (j = 0; j < p->len; j++)
		{
			//fprintf(stderr,"now writing at position %i, character %i\n", start_pos+j, p->seq[j] );
			write_to_half_byte_array(half_byte_array,start_pos+j,p->seq[j]);
		}
		//printf("index: %i\n", start_pos);
		start_pos += p->len;
	}
	return;
}

int copy_sequences_to_cuda_memory(
		bwa_seqio_t *bs,
		uint2 *global_sequences_index,
		uint2 *main_sequences_index,
		unsigned char *global_sequences,
		unsigned char *main_sequences,
		unsigned int *read_size,
		unsigned short *max_length,
		int *same_length,
		int buffer,
		unsigned long long *clump_array,
		unsigned char clump_len)
{
	//sum of length of sequences up the the moment
	unsigned int accumulated_length = 0;
	//sequence's read length

	unsigned int number_of_sequences = 0;

#define NEW_QUERY_READER 1

#if NEW_QUERY_READER == 0
error need to set same_length...
	unsigned short read_length = 0;
	while (bwa_read_seq_one_half_byte(bs,main_sequences,accumulated_length,&read_length,clump_array,clump_len,number_of_sequences)>0)
	{
		main_sequences_index[number_of_sequences].x = accumulated_length;
		main_sequences_index[number_of_sequences].y = read_length;
		if (read_length > *max_length) *max_length = read_length;

		accumulated_length += read_length;
		number_of_sequences++;

		if ( accumulated_length + MAX_SEQUENCE_LENGTH > (1ul<<(buffer+1)) ) break;
	}
#else
	int n_seqs = 0;
	barracuda_query_array_t *seqs = barracuda_read_seqs(bs,  buffer, &n_seqs, 0, 0, &accumulated_length, max_length, same_length);
	//TODO: insert  sort here!!!!
	//TODO: Arran: put the clumping code here.
	barracuda_write_to_half_byte_array(seqs, main_sequences, main_sequences_index, n_seqs);
	number_of_sequences = (unsigned int) n_seqs;

#endif

    //copy main_sequences_width from host to device
    hipUnbindTexture(sequences_index_array);
    report_cuda_error_GPU("[aln_core] Error freeing texture \"sequences_index_array\".");
    hipMemcpy(global_sequences_index, main_sequences_index, (number_of_sequences)*sizeof(uint2), hipMemcpyHostToDevice);
    report_cuda_error_GPU("[aln_core] Error copying to \"global_sequences_index\" on GPU");
    hipBindTexture(0, sequences_index_array, global_sequences_index, (number_of_sequences)*sizeof(uint2));
    report_cuda_error_GPU("[aln_core] Error binding texture \"sequences_index_array\".\n");

    //copy main_sequences from host to device, sequences array length should be accumulated_length/2
    hipUnbindTexture(sequences_array);
    report_cuda_error_GPU("[aln_core] Error freeing texture \"sequences_array\".");
    hipMemcpy(global_sequences, main_sequences, (1ul<<(buffer))*sizeof(unsigned char), hipMemcpyHostToDevice);
    report_cuda_error_GPU("[aln_core] Error copying to \"global_sequences\" on GPU");
    hipBindTexture(0, sequences_array, global_sequences, (1ul<<(buffer))*sizeof(unsigned char));
    report_cuda_error_GPU("[aln_core] Error binding texture to \"sequences_array\".\n");

    if ( read_size ) *read_size = accumulated_length;
    free (seqs);
    return number_of_sequences;
}

//CUDA DEVICE CODE STARTING FROM THIS LINE
/////////////////////////////////////////////////////////////////////////////

/*WBL 12 Feb 2015 performance much worse for barracuda r1.85
 * for timebeing try using full old version of bwt_cuda_occ4*/
#include "bwt_cuda_occ4.cuh"

#include "cuda.cuh"


/*WBL 11 feb 2015 dummy stub fix cuda_dfs_match() properly later**
__device__ ulong4 bwt_cuda_occ4(uint32_t *global_bwt, bwtint_t k) {
	int last = -1;		
#ifdef scache_global_bwt
	D_mycache;
#else
	//may need non-default __align__ to allow efficent access from __ldg()
#ifdef mycache4
	__align__(16) uint32_t mycache[size_mycache];
#else
#ifdef mycache2
	__align__(8) uint32_t mycache[size_mycache];
#else
	uint32_t mycache[size_mycache];
#endif
#endif
#endif //scache_global_bwt
  ulong4 n;
  n.x = bwt_cuda_occ(global_bwt, k, 0, 0, &last,l_mycache0);
  n.y = bwt_cuda_occ(global_bwt, k, 1, 0, &last,l_mycache0);
  n.z = bwt_cuda_occ(global_bwt, k, 2, 0, &last,l_mycache0);
  n.w = bwt_cuda_occ(global_bwt, k, 3, 0, &last,l_mycache0);
  return n;
}
*/

//configuration options for GP to tune
#undef direct_sequence
#include "bwt_cuda_match_exact.cuh"


#include "cuda2.cuh"
//END CUDA DEVICE CODE

// return the difference in second between two timeval structures
double diff_in_seconds(struct timeval *finishtime, struct timeval * starttime)
{
	double sec;
	sec=(finishtime->tv_sec-starttime->tv_sec);
	sec+=(finishtime->tv_usec-starttime->tv_usec)/1000000.0;
	return sec;
}

gap_opt_t *gap_init_bwaopt(gap_opt_t * opt)
{
	gap_opt_t *o;
	o = (gap_opt_t*)calloc(1, sizeof(gap_opt_t));
	o->s_mm = opt->s_mm;
	o->s_gapo = opt->s_gapo;
	o->s_gape = opt->s_gape;
	o->max_diff = opt->max_diff;
	o->max_gapo = opt->max_gapo;
	o->max_gape = opt->max_gape;
	o->indel_end_skip = opt->indel_end_skip;
	o->max_del_occ = opt->max_del_occ;
	o->max_entries = opt->max_entries;
	o->mode = opt->mode;
	o->seed_len = opt->seed_len;
	o->max_seed_diff = opt->max_seed_diff;
	o->fnr = opt->fnr;
	o->n_threads = 0;
	o->max_top2 = opt->max_top2;
	o->trim_qual = 0;
	return o;
}

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////

void add_to_process_queue(init_bin_list * bin, barracuda_aln1_t * aln, alignment_meta_t * partial, unsigned int sequence_id){
#if ARRAN_DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_core][split_bin] pos: %i k: %llu l: %llu mm: %i gapo: %i gape: %i score: %i seq: %i", partial->pos, aln->k, aln->l, aln->n_mm, aln->n_gapo, aln->n_gape, aln->score, sequence_id);
#endif
	init_list * new_aln = (init_list*) malloc(sizeof(init_list));
	memset(new_aln, 0, sizeof(init_list));
	if(bin->aln_list){
		new_aln->next = bin->aln_list;
	}
	bin->aln_list = new_aln;
	bin->processed = 0;

	init_info_t	*to_queue = &(new_aln->init);

	to_queue->score = aln->score;
	to_queue->sequence_id = sequence_id;
	to_queue->best_cnt = partial->best_cnt;

	//each kernel updates its own start_pos for the next round based upon the pass length it determines
	//this allows us to run partial hits from different points in sequences or for sequences of different lengths
	to_queue->start_pos = partial->pos;

	to_queue->lim_k = aln->k;
	to_queue->lim_l = aln->l;
	to_queue->cur_n_mm = aln->n_mm;
	to_queue->cur_n_gapo = aln->n_gapo;
	to_queue->cur_n_gape = aln->n_gape;
}

void stdout_aln_head(const int id, const int* no_of_alignments) {
#if STDOUT_STRING_RESULT == 1
  //output even if no_of_alignments <=0 
  printf("Sequence %d", id);
  printf(", no of alignments: %d\n", *no_of_alignments);
#else
  err_fwrite(no_of_alignments, 4, 1, stdout);
#endif
}

#if STDOUT_STRING_RESULT == 1
//ignore nmemb < MAX_NO_OF_ALIGNMENTS limit
#define stdout_aln1(type,opt_best_cnt)	\
  for(size_t i=0; i<nmemb; i++) {\
    printf("  Aligned read, ");\
    printf("n_mm: %d, ",   aln[i].n_mm);\
    printf("n_gape: %d, ", aln[i].n_gape);\
    printf("n_gapo: %d, ", aln[i].n_gapo);\
    printf("k: %llu, ",    aln[i].k);\
    printf("l: %llu, ",    aln[i].l);\
    printf("score: %d",    aln[i].score);\
    opt_best_cnt;\
    printf("\n");\
  }
#else
#define stdout_aln1(type,opt_best_cnt) \
  err_fwrite(aln, sizeof(type), nmemb, stdout);
#endif

void stdout_bwt_aln1(      const bwt_aln1_t       *aln, const size_t nmemb) {
  //fprintf(stderr,"stdout_bwt_aln1(*aln, %d) %dbytes\n",nmemb,sizeof(bwt_aln1_t));
  stdout_aln1(bwt_aln1_t,);
}
void stdout_barracuda_aln1(const barracuda_aln1_t *aln, const size_t nmemb) {
  //fprintf(stderr,"stdout_barracuda_aln1(*aln, %d) %dbytes\n",nmemb,sizeof(barracuda_aln1_t));
  stdout_aln1(barracuda_aln1_t,printf("best_cnt: %d", aln[i].best_cnt));
}
#undef stdout_aln1

/*WBL for debug
void print_global_alns(const int no_to_process, const int max_no_partial_hits, const barracuda_aln1_t * global_alns_device) {
  const size_t nbytes = max_no_partial_hits*no_to_process*sizeof(barracuda_aln1_t);
  barracuda_aln1_t * global_alns_host = (barracuda_aln1_t*)malloc(nbytes);
  assert(global_alns_host);
  hipMemcpy(global_alns_host, global_alns_device, nbytes, hipMemcpyDeviceToHost);
  report_cuda_error_GPU("[aln_core] Error reading \"global_alns_host\" from GPU for print.");

  for(int i=0;i<no_to_process;i++) {
    printf("alns %d ",i);
    for(int j=0;j<max_no_partial_hits;j++) {
      printf("%d %d %d ",
	     int(global_alns_host[i].n_mm),
	     int(global_alns_host[i].n_gapo),
	     int(global_alns_host[i].n_gape));
      printf("%lu %lu ",global_alns_host[i].k,global_alns_host[i].l);
      printf("%d %d",global_alns_host[i].score,global_alns_host[i].best_cnt);
      if(j<max_no_partial_hits-1) printf(", ");
    }
    printf("\n");
  }

  free(global_alns_host);
}*/
void core_kernel_loop(int sel_device, int buffer, gap_opt_t *opt, bwa_seqio_t *ks, double total_time_used, uint32_t *global_bwt)
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Core loop (this loads sequences to host memory, transfers to cuda device and aligns via cuda in CUDA blocks)
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
{
	//Variable Initializations
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//Timings
		double total_calculation_time_used = 0;
		struct timeval start, end;
		double time_used;

	//Sequence Reads related Variables
	// The length of the longest read;
		unsigned short max_sequence_length=0;
		// maximum read size from sequences in bytes
		int same_length;
		// Flag: are all sequences the same length
		unsigned int read_size = 0;
		// Number of sequences read per batchequences reside in global memory of GPU
		unsigned int no_of_sequences = 0;
		// total number of sequences read
		unsigned long long total_no_of_sequences = 0;
		// total number of reads in base pairs
		unsigned long long total_no_of_base_pair = 0;
		unsigned char * global_sequences = 0;
		// sequences reside in main memory of CPU
		unsigned char * main_sequences = 0;
		unsigned long long * main_suffixes = 0;
		// sequences index reside in global memory of GPU
		uint2 * global_sequences_index = 0;
		// sequences index reside in main memory of CPU
		uint2 * main_sequences_index = 0;

		// initializing pointer for device options from user
		gap_opt_t *options;

		// initial best score is the worst tolerated score without any alignment hit.
		const int best_score = aln_score(opt->max_diff+1, opt->max_gapo+1, opt->max_gape+1, opt);

		// global alignment stores for device
		//Variable for alignment result stores
		alignment_meta_t * global_alignment_meta_device;
		barracuda_aln1_t * global_alns_device;
		init_info_t * global_init_device;
		widths_bids_t * global_w_b_device;
		char * global_seq_flag_device, *global_seq_flag_host;
		// global alignment stores for host
		alignment_meta_t * global_alignment_meta_host, * global_alignment_meta_host_final;
		barracuda_aln1_t * global_alns_host, * global_alns_host_final;
		init_info_t * global_init_host;
#if USE_PETR_SPLIT_KERNEL > 0
		alignment_meta_t * global_alignment_meta_host_final;
#endif
		bwtkl_t * kl_device, *kl_host;



	//CPU and GPU memory allocations
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	gettimeofday (&start, NULL);
		//allocate global_sequences memory in device
		hipMalloc((void**)&global_sequences, (1ul<<(buffer))*sizeof(unsigned char));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_sequences\".");
		main_sequences = (unsigned char *)malloc((1ul<<(buffer))*sizeof(unsigned char));
		//suffixes for clumping
		main_suffixes = (unsigned long long *)malloc((1ul<<(buffer-3))*sizeof(unsigned long long));
		//allocate global_sequences_index memory in device assume the average length is bigger the 16bp (currently -3, -4 for 32bp, -3 for 16bp)long
		hipMalloc((void**)&global_sequences_index, (1ul<<(buffer-3))*sizeof(uint2));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_sequences_index\".");
		main_sequences_index = (uint2*)malloc((1ul<<(buffer-3))*sizeof(uint2));
		//allocate and copy options (opt) to device constant memory
		hipMalloc((void**)&options, sizeof(gap_opt_t));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"options\".");
		hipMemcpy ( options, opt, sizeof(gap_opt_t), hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error hipMemcpy to \"options\" on GPU");
		hipMemcpyToSymbol(HIP_SYMBOL( options_cuda), opt, sizeof(gap_opt_t), 0, hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error in hipMemcpyToSymbol to \"options_cuda\" on GPU");
		//allocate alignment stores for host and device
		hipMalloc((void**)&global_alignment_meta_device, (1ul<<(buffer-3))*sizeof(alignment_meta_t));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_alignment_meta_device\".");
		hipMalloc((void**)&global_alns_device, MAX_NO_PARTIAL_HITS*(1ul<<(buffer-3))*sizeof(barracuda_aln1_t));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_alns_device\".");
		hipMalloc((void**)&global_init_device, (1ul<<(buffer-3))*sizeof(init_info_t));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_init_device\".");
		hipMalloc((void**)&global_w_b_device, (1ul<<(buffer-3))*sizeof(widths_bids_t));
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_w_b_device\".");
		hipMalloc((void**)&global_seq_flag_device, (1ul<<(buffer-3))*sizeof(char));	
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"global_seq_flag_device\".");

		hipMalloc((void**)&kl_device, (1ul<<(buffer-3))*sizeof(bwtkl_t));	
		report_cuda_error_GPU("[core] Error allocating cuda memory for \"kl_device\".");

	//allocate alignment store memory in device assume the average length is bigger the 16bp (currently -3, -4 for 32bp, -3 for 16bp)long
		global_alignment_meta_host = (alignment_meta_t*)malloc((1ul<<(buffer-3))*sizeof(alignment_meta_t));
		assert(global_alignment_meta_host);//better than segfault later
		global_alns_host = (barracuda_aln1_t*)malloc(MAX_NO_PARTIAL_HITS*(1ul<<(buffer-3))*sizeof(barracuda_aln1_t));
		assert(global_alns_host);
		global_alignment_meta_host_final = (alignment_meta_t*)malloc((1ul<<(buffer-3))*sizeof(alignment_meta_t));
		assert(global_alignment_meta_host_final);
		global_alns_host_final = (barracuda_aln1_t*)malloc(MAX_NO_OF_ALIGNMENTS*(1ul<<(buffer-3))*sizeof(barracuda_aln1_t));
		assert(global_alns_host_final);
		global_init_host = (init_info_t*)malloc((1ul<<(buffer-3))*sizeof(init_info_t));
		assert(global_init_host);
		global_seq_flag_host = (char*)malloc((1ul<<(buffer-3))*sizeof(char));
		assert(global_seq_flag_host);
#if USE_PETR_SPLIT_KERNEL > 0
		global_alignment_meta_host_final = (alignment_meta_t*)malloc((1ul<<(buffer-3))*sizeof(alignment_meta_t));
		assert(global_alignment_meta_host_final);
#endif
		kl_host = (bwtkl_t*)malloc((1ul<<(buffer-3))*sizeof(bwtkl_t));
		assert(kl_host);

	gettimeofday (&end, NULL);
	time_used = diff_in_seconds(&end,&start);
	total_time_used += time_used;

#if DEBUG_LEVEL > 0
	fprintf(stderr,"[aln_debug] Finished allocating CUDA device memory\n");
#endif


	//Core loop starts here
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////

	gettimeofday (&start, NULL);
	int loopcount = 0;
	// unsigned int cur_sequence_id = 0; //unique sequence identifier
	// determine block size according to the compute capability
	int blocksize, avg_length;
	char split_engage;
	hipDeviceProp_t selected_properties;
	hipGetDeviceProperties(&selected_properties, sel_device);
	report_cuda_error_GPU("[core] Error on \"hipGetDeviceProperties\".");
	if ((int) selected_properties.major > 1) {
		blocksize = 64;
	} else {
		blocksize = 320;
	}

	while ( ( no_of_sequences = copy_sequences_to_cuda_memory(ks, global_sequences_index, main_sequences_index, global_sequences, main_sequences, &read_size, &max_sequence_length, &same_length, buffer, main_suffixes, SUFFIX_CLUMP_WIDTH) ) > 0 )
	{
		#define GRID_UNIT 32
		int gridsize = GRID_UNIT * (1 + int (((no_of_sequences/blocksize) + ((no_of_sequences%blocksize)!=0))/GRID_UNIT));
		dim3 dimGrid(gridsize);
		dim3 dimBlock(blocksize);

		avg_length = (read_size/no_of_sequences);
		split_engage = avg_length > SPLIT_ENGAGE;

		if(opt->seed_len > avg_length)
		{
			fprintf(stderr,"[aln_core] Warning! Specified seed length [%d] exceeds average read length, setting seed length to %d bp.\n", opt->seed_len, int ((read_size/no_of_sequences)>>1));
			opt->seed_len = read_size/no_of_sequences >> 1; //if specify seed length not valid, set to half the sequence length
		}

		if (!loopcount) fprintf(stderr, "[aln_core] Now aligning sequence reads to reference assembly, please wait..\n");

		if (!loopcount)	{
#if DEBUG_LEVEL > 0
			fprintf(stderr, "[aln_debug] Average read size: %dbp\n", read_size/no_of_sequences);
			fprintf(stderr, "[aln_debug] Using Reduced kernel\n");
			fprintf(stderr, "[aln_debug] Using SIMT with grid size: %u, block size: %d. ", gridsize,blocksize) ;
			fprintf(stderr,"\n[aln_debug] Loop count: %i\n", loopcount + 1);
#endif
			fprintf(stderr,"[aln_core] Processing %d sequence reads at a time.\n[aln_core] ", (gridsize*blocksize)) ;
		}
		//fprintf(stderr, "%d sequences max_sequence_length=%d same_length=%d\n", no_of_sequences, max_sequence_length, same_length);
		fprintf(stderr, "l%d", loopcount);
#if STDOUT_STRING_RESULT == 1
		fprintf(stdout, "loopcount %d\n", loopcount);
#endif

		gettimeofday (&end, NULL);
		time_used = diff_in_seconds(&end,&start);

		int run_no_sequences = no_of_sequences; //for compatibility to PETR_SPLIT_KERNEL only

		total_time_used+=time_used;

		// initialise the alignment stores
		memset(global_alignment_meta_host_final, 0, no_of_sequences*sizeof(alignment_meta_t));
		memset(global_alns_host_final, 0, no_of_sequences*MAX_NO_OF_ALIGNMENTS*sizeof(barracuda_aln1_t));
		for(int i=0; i<no_of_sequences; i++){
			global_alignment_meta_host_final[i].best_score = best_score;
		}

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Core match function per sequence readings
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		gettimeofday (&start, NULL);

#if DEBUG_LEVEL > 0
		fprintf(stderr,"\n[aln_debug] reduced kernel starts \n", time_used);
#endif
#if DEBUG_LEVEL > 3
		//printf("cuda opt:%d\n", cuda_opt);
#endif

		hipError_t cuda_err;

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Widths & Bids
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		//in this case, seq_flag is used to note sequences that have too many N characters
		cuda_prepare_widths<<<dimGrid,dimBlock>>>(global_bwt, no_of_sequences, global_w_b_device, global_seq_flag_device);
		//fprintf(stderr,"cuda_prepare_widths<<<(%d,%d,%d)(%d,%d,%d)>>>(global_bwt, %d, global_w_b_device, global_seq_flag_device)\n",
		//	dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_of_sequences);

		hipDeviceSynchronize();
		cuda_err = hipGetLastError();
		if(int(cuda_err))
		{
			fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported during width/bid preparation! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
			return;
		}
		/*if(loopcount==0){
		  const size_t nbytes = no_of_sequences*sizeof(widths_bids_t);
		  widths_bids_t* w_b = (widths_bids_t*)malloc(nbytes);
		  assert(w_b);
		  hipMemcpy(w_b, global_w_b_device, nbytes, hipMemcpyDeviceToHost);
		  report_cuda_error_GPU("[aln_core] Error reading \"global_w_b_device\" from GPU.");
		  for(int i=0;i<no_of_sequences;i++) {
		    printf("w_b %d ",i);
		    for(int j=0;j<max_sequence_length+1;j++) {
		      printf("%u %u",w_b[i].widths[j],int(w_b[i].bids[j]));
		      if(j<=max_sequence_length) printf(" ");
		    }
		    printf("\n");
		  }
		  free(w_b);
		}*/

		if(same_length) { /*new cuda_find_exact_matches assumes all sequences are same length*/
		//WBL re-enabled cuda_find_exact_matches with new KL output
		fprintf(stderr, "cuda_find_exact_matches<<<(%d,%d,%d)(%d,%d,%d)>>>(global_bwt, %d, %d, kl_device)\n",
			dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_of_sequences,max_sequence_length);
		struct timeval start2;
		gettimeofday (&start2, NULL);
		cuda_find_exact_matches<<<dimGrid,dimBlock>>>(global_bwt, no_of_sequences, max_sequence_length, kl_device);
		hipDeviceSynchronize();
		cuda_err = hipGetLastError();
		if(int(cuda_err))
		{
			fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported during exact match pre-check! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
			return;
		}
		gettimeofday (&end, NULL);
		const double time_used = diff_in_seconds(&end,&start2);
		fprintf(stderr, "[aln_core] find_exact_matches Kernel speed: %g sequences/sec or %g bp/sec %g\n", no_of_sequences/time_used, read_size/time_used, time_used);
		}
		hipMemcpy(kl_host, kl_device, no_of_sequences*sizeof(bwtkl_t), hipMemcpyDeviceToHost);
		report_cuda_error_GPU("[aln_core] Error reading \"kl_host\" from GPU.");

		/*if(loopcount==0)
		for(int i=0;i<no_of_sequences;i++) {
		  printf("kl %d %lu %lu\n",i,kl_host[i].k,kl_host[i].l);
		}*/

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Exclude exact unique matches and
		// Cull for too many Ns
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		memset(global_init_host, 0, no_of_sequences*sizeof(init_info_t));
		hipMemcpy(global_seq_flag_host, global_seq_flag_device, no_of_sequences*sizeof(char), hipMemcpyDeviceToHost);
		report_cuda_error_GPU("[aln_core] Error reading \"global_seq_flag_host\" from GPU.");
		hipDeviceSynchronize();
		report_cuda_error_GPU("[aln_core] cuda error");
		unsigned int no_to_process = 0;
		for(int i=0; i<no_of_sequences; i++){
		    //use K,L values to note sequences that have a unique exact match - allows setting of best_score=0 and skiping rest of processing
			if(same_length && /*ie cuda_find_exact_matches has been run*/
			   kl_host[i].k == kl_host[i].l) {
		    //save k and l, clear rest (n_mm etc)
				barracuda_aln1_t * tmp_aln = global_alns_host_final + i*MAX_NO_OF_ALIGNMENTS;
				memset(tmp_aln,0,sizeof(barracuda_aln1_t)); //clear n_mm, n_gapo,n_gape, score, best_cnt
				memcpy(&(tmp_aln->k),&kl_host[i].k,sizeof(bwtkl_t));
				//tmp_aln->n_mm = 100+loopcount; //for debug
		    //make sure sequence is marked so not processed again
				memset(global_alignment_meta_host_final + i, 0, sizeof(alignment_meta_t));		    
				global_alignment_meta_host_final[i].no_of_alignments = 1;
				//best_score = 0;
				global_alignment_meta_host_final[i].sequence_id = i;
				global_alignment_meta_host_final[i].best_cnt = 1;
				//char pos = 0;
				global_alignment_meta_host_final[i].finished = 1;
				//fprintf(stderr, "global_alignment_meta_host_final[%d].sequence_id = %d\n",i,global_alignment_meta_host_final[i].sequence_id);
			} else
			if(global_seq_flag_host[i]){
				memset(global_alignment_meta_host_final + i, 0, sizeof(alignment_meta_t));
				global_alignment_meta_host_final[i].sequence_id = i;
			}
			else {
				global_init_host[no_to_process].sequence_id = i;
				no_to_process++;
			}
		}

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Clump based on matching suffixes
		// Assumes that sequences were sorted
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		unsigned char first=1, clumping;
		unsigned int prev_suffix, old_no_to_process=no_to_process; //, unique=0;
		suffix_bin_list *suff_lst=0, *currSuff, *suff_ptrs[no_of_sequences];
		suffix_seq_list *currSeq;

		if(SUFFIX_CLUMP_WIDTH){
			for(int i=0; i<no_to_process; i++){
				if(main_suffixes[i]!=prev_suffix || first){
#if ARRAN_DEBUG_LEVEL > 0
	fprintf(stderr, "\n[aln_core][suffix_bin] Creating bin for suffix %llu", main_suffixes[i]);
#endif
					currSuff = (suffix_bin_list*) malloc(sizeof(suffix_bin_list));
					memset(currSuff, 0, sizeof(suffix_bin_list));
					if(suff_lst){
						currSuff->next = suff_lst;
					}
					suff_lst = currSuff;
					first = 0;
					prev_suffix = main_suffixes[i];
					//unique++;
				}

				suff_ptrs[global_init_host[i].sequence_id] = suff_lst;
				currSeq = (suffix_seq_list*) malloc(sizeof(suffix_seq_list));
				memset(currSeq, 0, sizeof(suffix_seq_list));
				currSeq->sequence_id = global_init_host[i].sequence_id;
				if(suff_lst->seq){
					currSeq->next = suff_lst->seq;
				}
#if ARRAN_DEBUG_LEVEL > 0
	fprintf(stderr, "\n[aln_core][suffix_bin] Adding sequence %i to bin for suffix %llu", currSeq->sequence_id, main_suffixes[i]);
#endif
				suff_lst->seq = currSeq;
			}

			no_to_process = 0;
			currSuff = suff_lst;
			do {
				global_init_host[no_to_process].sequence_id = currSuff->seq->sequence_id;
				no_to_process++;
				currSuff = currSuff->next;
			}
			while(currSuff);

			clumping = old_no_to_process!=no_to_process;

#if ARRAN_DEBUG_LEVEL > -1
	fprintf(stderr, "\n[aln_core][suffix_clump] Width: %i - Reduced to %i of %i (%0.2f%%)", SUFFIX_CLUMP_WIDTH, no_to_process, old_no_to_process, 100*(1-float(no_to_process)/float(no_of_sequences)));
#endif

		}

		fprintf(stderr, "'");
		hipMemcpy(global_init_device, global_init_host, no_to_process*sizeof(init_info_t), hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error copying \"global_init_host\" to GPU.");
		//cuda_find_exact_matches writes straight to global_init_device so we can launch the first kernel and then deal with global_seq_flag_device

		{struct timeval start2;
		gettimeofday (&start2, NULL);

		cuda_inexact_match_caller<<<dimGrid,dimBlock>>>(global_bwt, no_to_process, global_alignment_meta_device, global_alns_device, global_init_device, global_w_b_device, best_score, split_engage, SUFFIX_CLUMP_WIDTH>0);
		fprintf(stderr,"1 cuda_inexact_match_caller<<<(%d,%d,%d)(%d,%d,%d)>>>(,%d,,,,,,,%d)\n",
			dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_to_process, SUFFIX_CLUMP_WIDTH);
		fprintf(stderr, "'");

		//***EXACT MATCH CHECK***
		//store knowledge of an exact match to be copied into init struct during partial hit queueing
		//hipMemcpy(global_seq_flag_host, global_seq_flag_device, no_of_sequences*sizeof(char), hipMemcpyDeviceToHost);
		//report_cuda_error_GPU("[aln_core] Error reading \"global_seq_flag_host\" from GPU.");
		//for(int i=0; i<no_of_sequences; i++){
//				if(global_seq_flag_host[i]){
//					global_alignment_meta_host_final[i].has_exact = 1;
//				}
		//}


#if DEBUG_LEVEL > 0
		fprintf(stderr,"\n[aln_debug] kernel started, waiting for data... \n", time_used);
#endif
		// Did we get an error running the code? Abort if yes.
		hipDeviceSynchronize(); //wait until kernel has had a chance to report error
		cuda_err = hipGetLastError();
		if(int(cuda_err))
		{
			fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported during first kernel run! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
			return;
		}

#if DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_debug] Last CUDA error message: %s\n[aln_core]", hipGetErrorString(cuda_err));
#endif

		//Check time
		gettimeofday (&end, NULL);
		const double time_used = diff_in_seconds(&end,&start2);
		fprintf(stderr, "[aln_core] 1 inexact Kernel speed: %u %g sequences/sec %g\n", no_to_process, no_to_process/time_used, time_used);
		}
		time_used = diff_in_seconds(&end,&start);
		total_calculation_time_used += time_used;
		total_time_used += time_used;
		fprintf(stderr, ".");
		// query device for error

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// retrieve alignment information from CUDA device to host
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		gettimeofday (&start, NULL);

		unsigned int process_capacity = (unsigned int) (1ul<<(buffer-3));

		alignment_meta_t	* partial, * final;
		barracuda_aln1_t	* aln, * final_aln;
		int max_score = aln_score(opt->max_diff+1, opt->max_gapo+1, opt->max_gape+1, opt);
		init_bin_list		* bins[max_score]; //will be a sparse array but it's tiny so memory wasting is minimal
		for(int s=0; s<max_score; s++){
			bins[s] = 0;
		}

		bins[0] = (init_bin_list*) malloc(sizeof(init_bin_list));
		memset(bins[0], 0, sizeof(init_bin_list));

		int split_loop_count = 0;
		do {
			hipDeviceSynchronize();
			report_cuda_error_GPU("[aln_core] cuda error_2.");
			if(!split_loop_count){
				fprintf(stderr, "'");
			}
			hipMemcpy(global_alignment_meta_host, global_alignment_meta_device, no_to_process*sizeof(alignment_meta_t), hipMemcpyDeviceToHost);
			report_cuda_error_GPU("[aln_core] Error reading \"global_alignment_meta_host\" from GPU.");
			int max_no_partial_hits = (!split_loop_count ? MAX_NO_SEEDING_PARTIALS : MAX_NO_REGULAR_PARTIALS);
			hipMemcpy(global_alns_host, global_alns_device, max_no_partial_hits*no_to_process*sizeof(barracuda_aln1_t), hipMemcpyDeviceToHost);
			report_cuda_error_GPU("[aln_core] Error reading \"global_alns_host\" from GPU.");
			hipDeviceSynchronize();
			report_cuda_error_GPU("[aln_core] cuda error_3.");
			//if(loopcount==0) print_global_alns(max_no_partial_hits,no_to_process,global_alns_device);
			if(!split_engage) break;

#if ARRAN_DEBUG_LEVEL > 0
	fprintf(stderr, "\n[aln_core][split_loop]");
#endif

			fprintf(stderr, ":");
			for(int i=0; i<no_to_process; i++){
				partial = global_alignment_meta_host + i;
				final = global_alignment_meta_host_final + partial->sequence_id;
#if ARRAN_DEBUG_LEVEL > 0
	if(partial->no_of_alignments==0){
		init_info_t partial_init = global_init_host[i];
		fprintf(stderr, "\n[aln_core][split_null] pos: %i k: %llu l: %llu mm: %i gapo: %i gape: %i state: %i seq: %i", partial_init.start_pos, partial_init.lim_k, partial_init.lim_l, partial_init.cur_n_mm, partial_init.cur_n_gapo, partial_init.cur_n_gape, partial_init.cur_state, partial->sequence_id);
	}
#endif

				unsigned long long	partial_offset = i*max_no_partial_hits,
									final_offset = partial->sequence_id*MAX_NO_OF_ALIGNMENTS;
				for(int j=0; j<partial->no_of_alignments; j++){
					aln = global_alns_host + partial_offset + j;
					if(partial->finished){
						if(final->no_of_alignments==MAX_NO_OF_ALIGNMENTS){
							break;
						}
						final_aln = global_alns_host_final + final_offset + final->no_of_alignments;

						final_aln->k = aln->k;
						final_aln->l = aln->l;
						final_aln->n_mm = aln->n_mm;
						final_aln->n_gapo = aln->n_gapo;
						final_aln->n_gape = aln->n_gape;
						final_aln->score = aln->score;

#if ARRAN_DEBUG_LEVEL > 0
	fprintf(stderr, "\n[aln_core][split_complete] pos: %i k: %llu l: %llu mm: %i gapo: %i gape: %i score: %i", partial->pos, aln->k, aln->l, aln->n_mm, aln->n_gapo, aln->n_gape, aln->score);
#endif

						if(aln->score < final->best_score){
							final->best_score = aln->score;
							final->best_cnt = 0;
						}
						if(aln->score==final->best_score){
							final->best_cnt += aln->best_cnt;
						}

						final->no_of_alignments++;
					}
					else { // partial not finished

						//splice the linked list and add our new node in position to keep the ordering by score
						//keep a copy of the pointer for quick access
						if(!bins[aln->score]){
							init_bin_list	* prev, //used to find the bin with the greatest score < aln->score (i.e. previous one in the sorted list)
											* new_bin = (init_bin_list*) malloc(sizeof(init_bin_list));

							bins[aln->score] = new_bin;
							memset(new_bin, 0, sizeof(init_bin_list));
							new_bin->score = aln->score;

							for(prev = bins[0]; prev->next && prev->next->score < aln->score; prev = prev->next){}

							if(prev->next){
								new_bin->next = prev->next;
							}
#if ARRAN_DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_core][score_list] making bin: %i after %i", aln->score, prev->score);
#endif
							prev->next = new_bin;
						}

						if(SUFFIX_CLUMP_WIDTH && clumping && !split_loop_count){
							currSeq = suff_ptrs[partial->sequence_id]->seq;
							do {
								//NB note currSeq->sequence_id - not partial->sequence_id as with standard runs
								add_to_process_queue(bins[aln->score], aln, partial, currSeq->sequence_id);
								currSeq = currSeq->next;
							}
							while(currSeq);
						}
						else {
							add_to_process_queue(bins[aln->score], aln, partial, partial->sequence_id);
						}

					}
				}

			}

			fprintf(stderr, ":");
			init_bin_list * bin = bins[0];
			bool more_bins = true;
			int bins_processed=0, bins_to_process=split_loop_count<2 ? 1 : 2;
			for(no_to_process=0; more_bins && bins_processed<=bins_to_process && no_to_process<process_capacity; no_to_process++){
				while(!(bin->aln_list)){
					bins_processed += bin->processed;
					bin->processed = 0; //for the next loop
					if(!(bin->next) || bins_processed==bins_to_process){
						more_bins = false;
						break;
					}
					bin = bin->next;
				}
				if(!more_bins){
					break;
				}

				init_list * to_queue = bin->aln_list;

				final = global_alignment_meta_host_final + to_queue->init.sequence_id;
				//***EXACT MATCH CHECK***
				//to_queue->init.has_exact = final->has_exact;
				if(final->no_of_alignments){
					if(
							final->no_of_alignments==opt->max_aln
							|| to_queue->init.score > final->best_score + opt->s_mm //worst_tolerated_score will never be high enough
							|| (to_queue->init.score==final->best_score && final->best_cnt >= opt->max_top2) //best_cnt culling before it is even queued
					){ //woot! cull the tree!
#if ARRAN_DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_core][dfs_cull] init_score: %i	final_score: %i", to_queue->init.score, final->best_score);
#endif
						bin->aln_list = to_queue->next;
						free(to_queue);
						no_to_process--;
						continue;
					}
					to_queue->init.score = final->best_score;
				}
				else {
					to_queue->init.score = best_score; //give as much leeway as possible until alignments have been found and then cull the DFS tree
					to_queue->init.best_cnt = 0;
				}
				bin->processed = 1;

#if ARRAN_DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_core][split_queue] pos: %i k: %llu l: %llu mm: %i gapo: %i gape: %i score: %i bin: %i seq: %i", to_queue->init.start_pos, to_queue->init.lim_k, to_queue->init.lim_l, to_queue->init.cur_n_mm, to_queue->init.cur_n_gapo, to_queue->init.cur_n_gapo, bin->score, bins_processed, to_queue->init.sequence_id);
#endif

				memcpy(global_init_host + no_to_process, &(to_queue->init), sizeof(init_info_t));
				bin->aln_list = to_queue->next;
				free(to_queue);
			}

			if(no_to_process>0){
				fprintf(stderr, "|");
#if ARRAN_DEBUG_LEVEL > 0
		fprintf(stderr, "\n[aln_core][split_process] no_to_process: %i", no_to_process);
#endif
				hipMemcpy(global_init_device, global_init_host, no_to_process*sizeof(init_info_t), hipMemcpyHostToDevice);
				report_cuda_error_GPU("[aln_core] Error_2 copying \"global_init_host\" to GPU.");

				int gridsize = GRID_UNIT * (1 + int (((no_to_process/blocksize) + ((no_to_process%blocksize)!=0))/GRID_UNIT));
				dim3 dimGrid(gridsize);
				struct timeval start2;
				gettimeofday (&start2, NULL);
				cuda_inexact_match_caller<<<dimGrid,dimBlock>>>(global_bwt, no_to_process, global_alignment_meta_device, global_alns_device, global_init_device, global_w_b_device, best_score, split_engage, 0);
				fprintf(stderr,"2 cuda_inexact_match_caller<<<(%d,%d,%d)(%d,%d,%d)>>>(,%d,,,,,,,0)\n",
					dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_to_process);
				hipDeviceSynchronize(); //wait until kernel has had a chance to report error
				cuda_err = hipGetLastError();
				if(int(cuda_err))
				{
					fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported during split kernel run! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
					return;
				}
				//if(loopcount==0) print_global_alns(no_to_process,(!split_loop_count ? MAX_NO_SEEDING_PARTIALS : MAX_NO_REGULAR_PARTIALS),global_alns_device);
				gettimeofday (&end, NULL);
				const double time_used = diff_in_seconds(&end,&start2);
				fprintf(stderr, "[aln_core] 2 inexact Kernel speed: %u %g sequences/sec %g\n", no_to_process, no_to_process/time_used, time_used);
			}
			split_loop_count++;
		}
		while(no_to_process>0);

		//pop everything and free it
		init_bin_list_t * top = bins[0], * to_free;
		do {
			to_free = top;
			top = to_free->next;
			free(to_free);
		} while(top);

		if(SUFFIX_CLUMP_WIDTH){
			do {
				do {
					currSeq = suff_lst->seq;
					suff_lst->seq = suff_lst->seq->next;
					free(currSeq);
				}
				while(suff_lst->seq);
				currSuff = suff_lst;
				suff_lst = suff_lst->next;
				free(currSuff);
			}
			while(suff_lst);
		}

		if(split_engage){
			memcpy(global_alignment_meta_host, global_alignment_meta_host_final, no_of_sequences*sizeof(alignment_meta_t));
			memcpy(global_alns_host, global_alns_host_final, MAX_NO_OF_ALIGNMENTS*no_of_sequences*sizeof(barracuda_aln1_t));
		}


#if DEBUG_LEVEL > 0
		fprintf(stderr,"\n[aln_debug] Kernel finished, transfering data to host... \n", time_used);
#else
		fprintf(stderr,".");
#endif


#if USE_PETR_SPLIT_KERNEL > 0

		const int pass_length = (opt->seed_len > PASS_LENGTH)? opt->seed_len: PASS_LENGTH;
		const int split_engage = pass_length + 6;


		// which kernel are we running?
		char split_kernel = (read_size/no_of_sequences >= split_engage);

		split_kernel = 0;
#if DEBUG_LEVEL > 0
		fprintf(stderr,"[aln_debug] pass length %d, split engage %d.\n", pass_length, split_engage);
#endif

		if (!loopcount) fprintf(stderr, "[aln_core] Now aligning sequence reads to reference assembly, please wait..\n");

		if (!loopcount)	{
#if DEBUG_LEVEL > 0

			fprintf(stderr, "[aln_debug] Average read size: %dbp\n", read_size/no_of_sequences);

			if (split_kernel)
				fprintf(stderr, "[aln_debug] Using split kernel\n");
			else
				fprintf(stderr, "[aln_debug] Using normal kernel\n");
				fprintf(stderr,"[aln_core] Using SIMT with grid size: %u, block size: %d.\n[aln_core] ", gridsize,blocksize) ;
#endif


			fprintf(stderr,"[aln_core] Processing %d sequence reads at a time.\n[aln_core] ", (gridsize*blocksize)) ;
		}

		// zero out the final alignment store
		memset(global_alignment_meta_host_final, 0, (1ul<<(buffer-3))*sizeof(alignment_meta_t));

		// create host memory store which persists between kernel calls, on the stack
		main_alignment_store_host_t  main_store;
		memset(main_store.score_align, 0, MAX_SCORE*sizeof(align_store_lst *));


		run_no_sequences = no_of_sequences;
		gettimeofday (&end, NULL);
		time_used = diff_in_seconds(&end,&start);

		//fprintf(stderr,"time used: %f\n", time_used);

		total_time_used+=time_used;

		// initialise the alignment stores
		memset(global_alignment_meta_host, 0, (1ul<<(buffer-3))*sizeof(alignment_meta_t));

		for (int i = 0; i < no_of_sequences; i++)
		{
			alignment_meta_t* tmp = global_alignment_meta_host + i;

			// store the basic info to filter alignments into initialisation file
			tmp->init.lim_k = 0;
			tmp->init.lim_l = forward_seq_len;
			tmp->init.sequence_type = 0;
			tmp->start_pos = 0; //first part
			tmp->sequence_id = i; //cur_sequence_id; cur_sequence_id++;
			//if (!split_kernel) tmp->finished = 1;//mark as finished for normal kernel
		}

		// copy the initialised alignment store to the device
		hipMemcpy (global_alignment_meta_device,global_alignment_meta_host, no_of_sequences*sizeof(alignment_meta_t), hipMemcpyHostToDevice);
		report_cuda_error_GPU("[aln_core] Error copying \"global_alignment_meta_host\" to GPU.");

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Core match function per sequence readings
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		gettimeofday (&start, NULL);

#if DEBUG_LEVEL > 3
		printf("cuda opt:%d\n", cuda_opt);
#endif

		fprintf(stderr,"[aln_debug] kernels run \n", time_used);
		//debug only

		if (split_kernel) {
			assert(0);//should we be here?
			cuda_split_inexact_match_caller<<<dimGrid,dimBlock>>>(no_of_sequences, max_sequence_length, global_alignment_meta_device, 0);
			fprintf(stderr,"cuda_split_inexact_match_caller<<<(%d,%d,%d)(%d,%d,%d)>>>(%d, %d, global_alignment_meta_device, 0)\n",
				dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_of_sequences,max_sequence_length);
		} else {
			assert(0);//should we be here?
		  //WBL 21 Nov 2014 looks odd cuda_inexact_match_caller arguments do not match
			cuda_inexact_match_caller<<<dimGrid,dimBlock>>>(global_bwt, no_of_sequences, max_sequence_length, global_alignment_meta_device, 0);
			fprintf(stderr,"3 cuda_inexact_match_caller<<<(%d,%d,%d)(%d,%d,%d)>>>(global_bwt, %d, %d, global_alignment_meta_device, 0)\n",
				dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_of_sequences,max_sequence_length);
		}
		fprintf(stderr,"[aln_debug] kernels return \n", time_used);

		// Did we get an error running the code? Abort if yes.
		hipDeviceSynchronize(); //wait until kernel has had a chance to report error
		hipError_t cuda_err = hipGetLastError();
		if(int(cuda_err))
		{
			fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
			return;
		}

		//Check time
		gettimeofday (&end, NULL);
		time_used = diff_in_seconds(&end,&start);
		total_calculation_time_used += time_used;
		total_time_used += time_used;
		fprintf(stderr, ".");
		// query device for error

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// retrieve alignment information from CUDA device to host
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		gettimeofday (&start, NULL);

		char cont = 2;
		do
		{
			fprintf(stderr,".");
			hipMemcpy (global_alignment_meta_host, global_alignment_meta_device, no_of_sequences*sizeof(alignment_meta_t), hipMemcpyDeviceToHost);
			report_cuda_error_GPU("[aln_core] Error_2 reading \"global_alignment_meta_host\" from GPU.");

			// go through the aligned sequeces and decide which ones are finished and which are not
			int aligned=0;
			int alignments = 0;
			for (int i = 0; i < no_of_sequences; i++)
			{
				alignment_meta_t* tmp = global_alignment_meta_host + i;


				if (tmp->no_of_alignments > 0)
				{
					aligned += 1;
					alignments += tmp->no_of_alignments;
					//int seq_id = tmp->sequence_id;

					alignment_meta_t* final = global_alignment_meta_host_final + tmp->sequence_id;


					if (tmp->finished == 1 && final->no_of_alignments == 0) {
					// TODO debug seeding only
					//if (true) {
						memcpy(final, tmp, sizeof(alignment_meta_t)); //simply copy the alignment
#if DEBUG_LEVEL > 3
							printf("stored finished alignment for seq: %d\n", tmp->sequence_id);
#endif
					} else {
						// more processing needed, append if finished or enqueue otherwise
						for (int j = 0; j < tmp->no_of_alignments && j < MAX_NO_OF_ALIGNMENTS; j++)
						{
							if (tmp->finished == 1) {
								// append alignments to an existing entry
								int cur_no_aln = final->no_of_alignments;

								if (cur_no_aln + 1 < MAX_NO_OF_ALIGNMENTS) {
									final->alignment_info[cur_no_aln] = tmp->alignment_info[j];
									final->no_of_alignments = cur_no_aln + 1;
								} else {
									break;
								}

#if DEBUG_LEVEL > 3
								printf("stored finished alignment for seq: %d\n", tmp->sequence_id);
#endif
							} else {
#if DEBUG_LEVEL > 3
								printf("continue with unfinished seq: %d\n", tmp->sequence_id);
#endif
								// otherwise add them to another queue for processing
								int score = tmp->alignment_info[j].score;
								align_store_lst *cur_top = main_store.score_align[score];
								align_store_lst *new_top = (align_store_lst*) malloc( sizeof(align_store_lst) );

								new_top->val = tmp->alignment_info[j];
								new_top->sequence_id = tmp->sequence_id;
								new_top->next = cur_top;
								new_top->start_pos = tmp->start_pos;

								main_store.score_align[score] = new_top;
							}
						}
					}
				}
			}

			#if DEBUG_LEVEL > 0

			fprintf(stderr, "[aln_debug] seq. through: %i \n", aligned);
			fprintf(stderr, "[aln_debug] total alignments: %i \n", alignments);

			#endif

			//print out current new host alignment store
#if DEBUG_LEVEL > 3
			for (int j=0; j<MAX_SCORE; j++) {
					align_store_lst * cur_el = main_store.score_align[j];

					if (cur_el) {
						printf("Alignments with score: %d \n", j);
					}

					while(cur_el) {
						barracuda_aln1_t alignment = cur_el->val;
						int cur_len = main_sequences_index[cur_el->sequence_id].y;
						//print some info
						printf("Sequence: %d,  a:%d, k: %d, l: %d, mm: %d, gape: %d, gapo: %d, length: %d, processed: %d\n",cur_el->sequence_id, alignment.a, alignment.k, alignment.l, alignment.n_mm, alignment.n_gape, alignment.n_gapo, cur_len, cur_el->start_pos);

						cur_el = cur_el->next;
					}


			}
			printf("\n");
#endif



			int max_process = (1ul<<(buffer-3)); //taken from the line allocating the memory, maximum we can do in a single run

			int last_index = -1;


			//remove items from the list and free memory accordingly
			for(int i=0; i<MAX_SCORE && max_process > last_index+1; i++) {
				align_store_lst * cur_el = main_store.score_align[i];
				align_store_lst * tmp;

				while(cur_el  && max_process > last_index+1) {
					barracuda_aln1_t alignment = cur_el->val;


					// add alignment to the new store
					last_index++;
					alignment_meta_t* store_entry = global_alignment_meta_host + (last_index);

					// increment start_pos
					store_entry->start_pos = cur_el->start_pos + pass_length;

					store_entry->sequence_id = cur_el->sequence_id;
	//				store_entry->init.best_cnt = alignment.best_cnt;
	//				store_entry->init.best_diff = alignment.best_diff;
					store_entry->init.cur_n_gape = alignment.n_gape;
					store_entry->init.cur_n_gapo = alignment.n_gapo;
					store_entry->init.cur_n_mm = alignment.n_mm;
					store_entry->init.lim_k = alignment.k;
					store_entry->init.lim_l = alignment.l;
					store_entry->init.sequence_type = alignment.a;
					store_entry->no_of_alignments = 0; //change to 1 to see the prev. alignment

					tmp = cur_el;
					cur_el = cur_el->next;

					// update the main store to point to the new element
					main_store.score_align[i] = cur_el;

					free(tmp);
				}

			}

			no_of_sequences = last_index + 1;


			if (no_of_sequences > 0) {

#if DEBUG_LEVEL > 3
				printf("aligning %d sequences\n", no_of_sequences);
#endif

				// how many blocks in the current run
				gridsize = GRID_UNIT * (1 + int (((no_of_sequences/blocksize) + ((no_of_sequences%blocksize)!=0))/GRID_UNIT));
				dimGrid = gridsize;

				// transfer the data to the card again
				hipMemcpy (global_alignment_meta_device,global_alignment_meta_host, no_of_sequences*sizeof(alignment_meta_t), hipMemcpyHostToDevice);
				report_cuda_error_GPU("[aln_core] Error_2 copying \"global_alignment_meta_host\" to GPU.");

				//run kernel again
				assert(0);//should we be here?
				cuda_split_inexact_match_caller<<<dimGrid,dimBlock>>>(no_of_sequences, max_sequence_length, global_alignment_meta_device, 0);
				fprintf(stderr,"cuda_split_inexact_match_caller<<<(%d,%d,%d)(%d,%d,%d)>>>(%d, %d, global_alignment_meta_device, 0)\n",
					dimGrid.x,dimGrid.y,dimGrid.z,dimBlock.x,dimBlock.y,dimBlock.z,no_of_sequences, max_sequence_length);

				// Did we get an error running the code? Abort if yes.
				hipDeviceSynchronize(); //wait until kernel has had a chance to report error
				hipError_t cuda_err = hipGetLastError();
				if(int(cuda_err))
				  {
				    fprintf(stderr, "\n[aln_core] CUDA ERROR(s) reported! Last CUDA error message: %s\n[aln_core] Abort!\n", hipGetErrorString(cuda_err));
				    return;
				  }

			}
			else {
#if DEBUG_LEVEL > 3
				printf("Nothing to align, finished \n");
#endif
				cont = 0;
			}

		} while(cont);
		// end of kernel loop

#endif

#if DEBUG_LEVEL > 0
		if(opt->bwa_output)
			fprintf(stderr,"[aln_debug] Writing alignment to disk in BWA compatible format...");
		else
			fprintf(stderr,"[aln_debug] Writing alignment to disk in old barracuda format...");
#endif
		for (int  i = 0; i < run_no_sequences; ++i)
		{
#if USE_PETR_SPLIT_KERNEL > 0
			alignment_meta_t* tmp = global_alignment_meta_host_final + i;
#else
			alignment_meta_t* tmp = global_alignment_meta_host + i;
#endif
			stdout_aln_head(i,&tmp->no_of_alignments);
			if (tmp->no_of_alignments)
			{
				unsigned long long aln_offset = i*MAX_NO_OF_ALIGNMENTS;
				barracuda_aln1_t * tmp_aln;
				if(opt->bwa_output)
				{
					bwt_aln1_t * output;
					output = (bwt_aln1_t*)malloc(tmp->no_of_alignments*sizeof(bwt_aln1_t));
					memset(output,0,tmp->no_of_alignments*sizeof(bwt_aln1_t));//avoid undefined bytes in .sai files

					for (int j = 0; j < tmp->no_of_alignments; j++)
					{
						tmp_aln = global_alns_host_final + aln_offset + j;
						bwt_aln1_t * temp_output = output + j;
						//temp_output->a = tmp->alignment_info[j].a;
						temp_output->k = tmp_aln->k;
						temp_output->l = tmp_aln->l;
						temp_output->n_mm = tmp_aln->n_mm;
						temp_output->n_gapo = tmp_aln->n_gapo;
						temp_output->n_gape = tmp_aln->n_gape;
						temp_output->score = tmp_aln->score;
					}
					if(tmp->no_of_alignments > 1) aln_quicksort(output,0,tmp->no_of_alignments-1);
					stdout_bwt_aln1(output, tmp->no_of_alignments);
					free(output);
				}else
				{
					barracuda_aln1_t * output;
					output = (barracuda_aln1_t*)malloc(tmp->no_of_alignments*sizeof(barracuda_aln1_t));
					memset(output,0,tmp->no_of_alignments*sizeof(barracuda_aln1_t)); //avoid undefined bytes in .sai files

					for (int j = 0; j < tmp->no_of_alignments; j++)
					{
						tmp_aln = global_alns_host_final + aln_offset + j;
						barracuda_aln1_t * temp_output = output + j;
						//temp_output->a = tmp_aln->a;
						temp_output->k = tmp_aln->k;
						temp_output->l = tmp_aln->l;
						temp_output->n_mm = tmp_aln->n_mm;
						temp_output->n_gapo = tmp_aln->n_gapo;
						temp_output->n_gape = tmp_aln->n_gape;
						temp_output->score = tmp_aln->score;
					}
					stdout_barracuda_aln1(output, tmp->no_of_alignments);
					free(output);
				}
			}
		}

		gettimeofday (&end, NULL);
		time_used = diff_in_seconds(&end,&start);
		total_time_used += time_used;
		//fprintf(stderr, "Finished outputting alignment information... %0.2fs.\n\n", time_used);
		fprintf (stderr, ".");
		total_no_of_base_pair+=read_size;
		total_no_of_sequences+=run_no_sequences;
		gettimeofday (&start, NULL);
		loopcount ++;
	}
	fprintf(stderr, "\n");

	//report if there is any CUDA error
	hipError_t cuda_err = hipGetLastError();
	if(int(cuda_err))
	{
		fprintf(stderr, "[aln_core] CUDA ERROR(s) reported at end of core loop! Message: %s\n", hipGetErrorString(cuda_err));
	}

#if DEBUG_LEVEL > 0
	fprintf(stderr, "[aln_debug] ERROR message: %s\n", hipGetErrorString( hipGetLastError() ) );
#endif

	fprintf(stderr, "[aln_core] Finished!\n[aln_core] Total no. of sequences: %u, size in base pair: %u bp, average length %0.2f bp/sequence.\n", (unsigned int)total_no_of_sequences, (unsigned int)total_no_of_base_pair, (float)total_no_of_base_pair/(unsigned int)total_no_of_sequences);
	fprintf(stderr, "[aln_core] Alignment Speed: %0.2f sequences/sec or %0.2f bp/sec.\n", (float)(total_no_of_sequences/total_time_used), (float)(total_no_of_base_pair/total_time_used));
	fprintf(stderr, "[aln_core] Total program time: %0.2fs.\n", (float)total_time_used);

	//Free memory
	hipFree(global_sequences);
	free(main_sequences);
	free(main_suffixes);
	hipFree(global_sequences_index);
	hipFree(kl_device);
	free(main_sequences_index);
	free(kl_host);
	hipFree(global_alignment_meta_device);
	hipFree(global_alns_device);
	hipFree(global_seq_flag_device);
	free(global_alignment_meta_host);
	free(global_alns_host);
	free(global_alignment_meta_host_final);
	free(global_alns_host_final);
	free(global_seq_flag_host);

	return;
}


void cuda_alignment_core(const char *prefix, bwa_seqio_t *ks,  gap_opt_t *opt)
//Determines the availability of CUDA devices and
//calls core_kernel_loop();

//invokes CUDA kernels cuda_inexact_match_caller
{
	// For timing purpose only
	struct timeval start, end;
	double time_used;
	double total_time_used = 0;


	fprintf(stderr,"[aln_core] Running %s CUDA mode.\n",PACKAGE_VERSION);
#if STDOUT_STRING_RESULT == 1
	fprintf(stdout,"[aln_core] Running %s CUDA mode.\n",PACKAGE_VERSION);
#endif
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//CUDA options
	if (opt->max_entries < 0 )
	{
		opt->max_entries = 150000;
	}

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Pick Cuda device
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////

	size_t mem_available = 0, total_mem = 0; //, max_mem_available = 0;
	hipDeviceProp_t properties;
	int num_devices;
	hipGetDeviceCount(&num_devices);
	report_cuda_error_GPU("[core] Error on hipGetDeviceCount");

	if (!num_devices)
	{
		fprintf(stderr,"[aln_core] Cannot find a suitable CUDA device! aborting!\n");
	}


	int sel_device = 0;
	if (opt->cuda_device == -1)
	{
		sel_device = detect_cuda_device();
		if(sel_device >= 0)
		{
			hipSetDevice(sel_device);
			report_cuda_error_GPU("[core] Error on hipSetDevice");
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			report_cuda_error_GPU("[core] Error on hipFuncCachePreferL1");
		}
		else
		{
			fprintf(stderr,"[aln_core] Cannot find a suitable CUDA device! aborting!\n");
			return;
		}
	}
	else if (opt->cuda_device >= 0)
	{
		 sel_device = opt->cuda_device;
		 hipSetDevice(sel_device);
		 report_cuda_error_GPU("[core] Error_2 on hipSetDevice");
		 hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		 report_cuda_error_GPU("[core] Error_2 on hipFuncCachePreferL1");
		 hipGetDeviceProperties(&properties, sel_device);
		 report_cuda_error_GPU("[core] Error on hipGetDeviceProperties");
		 hipMemGetInfo(&mem_available, &total_mem);
		 report_cuda_error_GPU("[core] Error on hipMemGetInfo");

		 fprintf(stderr, "[aln_core] Using specified CUDA device %d, memory available %d MB.\n", sel_device, int(mem_available>>20));

	}

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy bwt occurrences array to from HDD to CPU then to GPU
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////

	gettimeofday (&start, NULL);
	bwtint_t seq_len;

	// pointer to bwt occurrence array in GPU
	uint32_t * global_bwt = 0;
	// total number of bwt_occ structure read in bytes
	unsigned long long bwt_read_size = 0;

	hipMemGetInfo(&mem_available, &total_mem);
		 report_cuda_error_GPU("[core] Error_2 on hipMemGetInfo");
	fprintf(stderr,"[aln_core] Loading BWTs, please wait..\n");

	bwt_read_size = copy_bwts_to_cuda_memory(prefix, &global_bwt, mem_available>>20, &seq_len)>>20;

	// copy_bwt_to_cuda_memory
	// returns 0 if error occurs
	// mem_available in MiB not in bytes

	if (!bwt_read_size) return; //break

	gettimeofday (&end, NULL);
	time_used = diff_in_seconds(&end,&start);
	total_time_used += time_used;
	fprintf(stderr, "[aln_core] Finished loading reference sequence assembly, %u MB in %0.2fs (%0.2f MB/s).\n", (unsigned int)bwt_read_size, time_used, ((unsigned int)bwt_read_size)/time_used );


	///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// allocate GPU working memory
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//Set memory buffer according to memory available
	hipMemGetInfo(&mem_available, &total_mem);
	report_cuda_error_GPU("[core] Error_3 on hipMemGetInfo");

	#if DEBUG_LEVEL > 0
	fprintf(stderr,"[aln_debug] mem left: %d MiB\n", int(mem_available>>20));
	#endif

	//stop if there isn't enough memory available

	int buffer = SEQUENCE_TABLE_SIZE_EXPONENTIAL;
	if ((mem_available>>20) < CUDA_TESLA)
	{
		buffer = buffer - 1; //this will half the memory usage by half to 675MB
		if(mem_available>>20 < (CUDA_TESLA >> 1))
		{
			fprintf(stderr,"[aln_core] Not enough memory to perform alignment (min: %d MB).\n", CUDA_TESLA >> 1);
			return;
		}
	}
	else
	{
		fprintf(stderr,"[aln_core] Sweet! Running with an enlarged buffer.\n");
	}

	//calls core_kernel_loop
	core_kernel_loop(sel_device, buffer, opt, ks, total_time_used, global_bwt);

	free_bwts_from_cuda_memory(global_bwt);

	return;
}

//////////////////////////////////////////
// CUDA detection code
//////////////////////////////////////////

int bwa_deviceQuery()
// Detect CUDA devices available on the machine, for quick CUDA test and for multi-se/multi-pe shell scripts
{
	int device, num_devices;
	hipGetDeviceCount(&num_devices);
	report_cuda_error_GPU("[core] Error_2 on hipGetDeviceCount");
	if (num_devices)
		{
			  //fprintf(stderr,"[deviceQuery] Querying CUDA devices:\n");
			  for (device = 0; device < num_devices; device++)
			  {
					  hipDeviceProp_t properties;
					  hipGetDeviceProperties(&properties, device);
					  report_cuda_error_GPU("[core] Error_3 on hipGetDeviceProperties");
					  fprintf(stdout, "%d ", device);
					  fprintf(stdout, "%d %d%d\n", int(properties.totalGlobalMem>>20), int(properties.major),  int(properties.minor));

			  }
			  //fprintf(stderr,"[total] %d\n", device);
		}
	return 0;
}

int detect_cuda_device()
// Detect CUDA devices available on the machine, used in aln_core and samse_core
{
	int num_devices, device = 0;
	size_t mem_available = 0,
		   //total_mem = 0,
		   max_mem_available = 0;
	hipGetDeviceCount(&num_devices);
	report_cuda_error_GPU("[detect_cuda_device] Error_3 on hipGetDeviceCount");
	hipDeviceProp_t properties;
	int sel_device = -1;

	if (num_devices >= 1)
	{
	     fprintf(stderr, "[detect_cuda_device] Querying CUDA devices:\n");
		 int max_cuda_cores = 0, max_device = 0;
		 for (device = 0; device < num_devices; device++)
		 {
			  hipGetDeviceProperties(&properties, device);
			  report_cuda_error_GPU("[detect_cuda_device] Error_4 on hipGetDeviceCount");
			  mem_available = properties.totalGlobalMem;
			  //hipMemGetInfo(&mem_available, &total_mem);
			  fprintf(stderr, "[detect_cuda_device]   Device %d ", device);
			  fprintf(stderr,"%s ", properties.name);
			  //calculated by multiprocessors * 8 for 1.x and multiprocessors * 32 for 2.0, *48 for 2.1 and *192 for 3.0
			  //determine amount of memory available
				  int cuda_cores = 0;
				  if (properties.major == 1){
						  cuda_cores = properties.multiProcessorCount*8;

				  }else if (properties.major == 2){
					  if (properties.minor == 0){
						  cuda_cores = properties.multiProcessorCount*32;
					  }else{
						  cuda_cores = properties.multiProcessorCount*48;
					  }
				  }else if (properties.major ==3)
				  {
					  cuda_cores = properties.multiProcessorCount*192;
				  }

			  fprintf(stderr,", CUDA cores %d, global memory size %d MB, compute capability %d.%d.\n", int(cuda_cores), int(mem_available>>20), int(properties.major),  int(properties.minor));
			  if (max_cuda_cores <= cuda_cores) //choose the one with highest number of processors
			  {
					  max_cuda_cores = cuda_cores;
					  if (max_mem_available < mem_available) //choose the one with max memory
					  {
						      max_mem_available = mem_available;
							  max_device = device;
					  }
			  }
 		 }
		 if (max_mem_available>>20 >= MIN_MEM_REQUIREMENT)
		 {
			 sel_device = max_device;
			 fprintf(stderr, "[detect_cuda_device] Using CUDA device %d, global memory size %d MB.\n", max_device, int(max_mem_available>>20));
			 }
		 else
		 {
			 fprintf(stderr,"[detect_cuda_device] Cannot find a suitable CUDA device with > %d MB of memory available! aborting!\n", MIN_MEM_REQUIREMENT);
			 return -1;
		 }
	}
	else
	{
		 fprintf(stderr,"[detect_cuda_device] No CUDA device found! aborting!\n");
		 return -1;
	}
	return sel_device;
}

//////////////////////////////////////////
// End CUDA detection code
//////////////////////////////////////////

//////////////////////////////////////////
// Below is code for BarraCUDA CUDA samse_core
//////////////////////////////////////////
#if CUDA_SAMSE == 1



void report_cuda_error_CPU(const char * message)
{
	fprintf(stderr,"%s\n",message);
	exit(1);
}


// Texture.
texture<bwtint_t, 1, hipReadModeElementType> sa_tex;
texture<bwtint_t, 1, hipReadModeElementType> bwt_sa_tex;
texture<bwtint_t, 1, hipReadModeElementType> rbwt_sa_tex;
texture<int, 1, hipReadModeElementType> g_log_n_tex;

// Variables for information to do with GPU or software (e.g., no. of blocks).

const static int BLOCK_SIZE2 = 128;

static bwa_maxdiff_mapQ_t *seqs_maxdiff_mapQ_ho;
static bwa_maxdiff_mapQ_t *seqs_maxdiff_mapQ_de;
static bwtint_t *seqs_sa_ho;
static bwtint_t *seqs_sa_de;
static uint8_t *seqs_mapQ_ho;
static uint8_t *seqs_mapQ_de;
static bwtint_t *seqs_pos_ho;
static bwtint_t *seqs_pos_de;


int prepare_bwa_cal_pac_pos_cuda1(
    unsigned int **global_bwt,
    unsigned int **global_rbwt,
    const char *prefix,
    bwtint_t **bwt_sa_de,
    bwtint_t **rbwt_sa_de,
    const bwt_t *bwt,
    const bwt_t *rbwt,
    const int *g_log_n_ho,
    int **g_log_n_de,
    const int g_log_n_len,
    int device)
{
    // mem_available in bytes not MiB
    size_t mem_available,total_mem;

    hipSetDevice(device);
    report_cuda_error_GPU("[samse_core] Error on hipSetDevice");
    hipMemGetInfo(&mem_available, &total_mem);
    report_cuda_error_GPU("[samse_core] Error on hipMemGetInfo");

    ////////////////////////////////////////////////////////////
    // Load BWT to GPU.
    ////////////////////////////////////////////////////////////

    // copy_bwt_occ_array_to_cuda_memory

	unsigned long long size_read = 0;

		if ( bwt != 0 )
		{
			//Original BWT
			size_read += bwt->bwt_size*sizeof(uint32_t);

			mem_available = mem_available - size_read;

			if(mem_available > 0)
			{
				//Allocate memory for bwt
				hipMalloc((void**)global_bwt, bwt->bwt_size*sizeof(uint32_t));
				report_cuda_error_GPU("[samse_core] Error allocating memory for \"global_bwt\"");
				//copy bwt occurrence array from host to device and dump the bwt to save CPU memory
				hipMemcpy (*global_bwt, bwt->bwt, bwt->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
				report_cuda_error_GPU("[samse_core] Error copying \"bwt\" to GPU.\n");
				//bind global variable bwt to texture memory bwt_occ_array
				hipBindTexture(0, bwt_occ_array, *global_bwt, bwt->bwt_size*sizeof(uint32_t));
				report_cuda_error_GPU("[samse_core] Error binding texture \"bwt_occ_array\".\n");
				//copy bwt structure data to constant memory bwt_cuda structure
				hipMemcpyToSymbol(HIP_SYMBOL( bwt_cuda), bwt, sizeof(bwt_t), 0, hipMemcpyHostToDevice);
				report_cuda_error_GPU("[samse_core] Error \"copy bwt\" to GPU constant memory");
			}
			else
			{
				fprintf(stderr,"[samse_core] Not enough device memory to continue.\n");
				return 0;
			}


	#if DEBUG_LEVEL > 0
			fprintf(stderr,"[samse_debug] bwt loaded, mem left: %d MB\n", (int)(mem_available>>20));
	#endif
		}
		if ( rbwt != 0 )
		{
			//Reversed BWT
			size_read += bwt->bwt_size*sizeof(uint32_t);
			mem_available = mem_available - (bwt->bwt_size*sizeof(uint32_t));


	#if DEBUG_LEVEL > 0
			fprintf(stderr,"[samse_debug] rbwt loaded mem left: %d MB\n", (int)(mem_available>>20));
	#endif

			if (mem_available > 0)
			{
				//Allocate memory for rbwt
				hipMalloc((void**)global_rbwt, rbwt->bwt_size*sizeof(uint32_t));
				report_cuda_error_GPU("[samse_core] Error on allocating cuda memory for \"global_rbwt\"");
				//copy reverse bwt occurrence array from host to device and dump the bwt to save CPU memory
				hipMemcpy (*global_rbwt, rbwt->bwt, rbwt->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
				report_cuda_error_GPU("[samse_core] Error copying rbwt->bwt to GPU.");
				//bind global variable rbwt to texture memory rbwt_occ_array
				hipBindTexture(0, rbwt_occ_array, *global_rbwt, rbwt->bwt_size*sizeof(uint32_t));
				report_cuda_error_GPU("[samse_core] Error binding texture \"rbwt_occ_array\".");
				//copy rbwt structure data to constant memory bwt_cuda structure
				hipMemcpyToSymbol(HIP_SYMBOL( rbwt_cuda), rbwt, sizeof(bwt_t), 0, hipMemcpyHostToDevice);
				report_cuda_error_GPU("[samse_core] Error copying \"rbwt\" to GPU constant memory");
			}
			else
			{
				fprintf(stderr,"[samse_core] Not enough device memory to continue.\n");
				return 0;
			}

		}

	// returns 0 if error occurs

    ////////////////////////////////////////////////////////////
    // Copy input data in "g_log_n" to device, and bind texture of "g_log_n_de".
    ////////////////////////////////////////////////////////////
    // Reserve memory.
    hipMalloc((void**)g_log_n_de,sizeof(int)*g_log_n_len);
    report_cuda_error_GPU("[samse_core] Error reserving memory for \"g_log_n_de\".\n");

    // Copy data from host to device.
    hipMemcpy(*g_log_n_de,g_log_n_ho,sizeof(int)*g_log_n_len,hipMemcpyHostToDevice);
    report_cuda_error_GPU("[samse_core] Error copying to \"g_log_n_de\".\n");

    // Bind texture.
    hipBindTexture(0,g_log_n_tex,*g_log_n_de,sizeof(int)*g_log_n_len);
    report_cuda_error_GPU("[samse_core] Error binding texture to \"g_log_n_tex\".\n");

    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // Copy "sa" data of BWT and RBWT to device.
    ///////////////////////////////////////////////////////////////////////////////////////////////////

    //fprintf(stderr,"[debug] bwt->n_sa: %u\n", bwt->n_sa);

    // Reserve memory for SA (BWT) on device.
    hipMalloc(*&bwt_sa_de,sizeof(bwtint_t)*bwt->n_sa);
    report_cuda_error_GPU("Error reserving memory for \"bwt_sa_de\".\n");

    // Copy SA (BWT) to device.
    hipMemcpy(*bwt_sa_de,bwt->sa,sizeof(bwtint_t)*bwt->n_sa,hipMemcpyHostToDevice);
    report_cuda_error_GPU("[samse_core] Error copying to \"bwt_sa_de\".\n");

    // Bind texture.
    hipBindTexture(0,bwt_sa_tex,*bwt_sa_de,sizeof(bwtint_t)*bwt->n_sa);
    report_cuda_error_GPU("[samse_core] Error binding texture to \"bwt_sa_tex\".\n");

    // Reserve memory for SA (RBWT) on device.
    hipMalloc(*&rbwt_sa_de,sizeof(bwtint_t)*rbwt->n_sa);
    report_cuda_error_GPU("[samse_core] Error reserving memory for \"rbwt_sa_de\".\n");

    // Copy SA (RBWT) to device.
    hipMemcpy(*rbwt_sa_de,rbwt->sa,sizeof(bwtint_t)*rbwt->n_sa,hipMemcpyHostToDevice);
    report_cuda_error_GPU("[samse_core] Error copying to \"rbwt_sa_de\".\n");

    // Bind texture.
    hipBindTexture(0,rbwt_sa_tex,*rbwt_sa_de,sizeof(bwtint_t)*rbwt->n_sa);
    report_cuda_error_GPU("[samse_core] Error binding texture to \"rbwt_sa_tex\".\n");

	hipMemGetInfo(&mem_available, &total_mem);
#if DEBUG_LEVEL > 0
	fprintf(stderr,"[samse_debug] sa/rsa loaded mem left: %d MB\n", (int)(mem_available>>20));
#endif
    return 1;
}


void prepare_bwa_cal_pac_pos_cuda2(int n_seqs_max)
{
	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Allocate memory and copy reads in "seqs" to "seqs_maxdiff_mapQ_ho" and "seqs_sa_ho".
	///////////////////////////////////////////////////////////////////////////////////////////////////
	seqs_maxdiff_mapQ_ho = (bwa_maxdiff_mapQ_t *) malloc(sizeof(bwa_maxdiff_mapQ_t)*n_seqs_max);
	if (seqs_maxdiff_mapQ_ho == NULL) report_cuda_error_CPU("[samse_core] Error reserving memory for \"seqs_maxdiff_mapq_ho\".\n");
	seqs_sa_ho = (bwtint_t *) malloc(sizeof(bwtint_t)*n_seqs_max);
	if (seqs_sa_ho == NULL) report_cuda_error_CPU("[samse_core] Error reserving memory for \"seqs_sa_ho\".\n");

    ///////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy input data in "seqs_maxdiff_mapQ_ho" and "seqs_sa_ho" to device, and bind texture of
	// "seqs_sa_de".
	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Reserve memory.
	hipMalloc(&seqs_maxdiff_mapQ_de,sizeof(bwa_maxdiff_mapQ_t)*n_seqs_max);
	report_cuda_error_GPU("Error reserving memory for \"seqs_maxdiff_mapQ_de\".\n");

	// Reserve memory.
	hipMalloc(&seqs_sa_de,sizeof(bwtint_t)*n_seqs_max);
	report_cuda_error_GPU("Error reserving memory for \"seqs_sa_de\".\n");

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Reserve memory for output data variables in "seqs_mapQ_ho" and "seqs_pos_ho" to host.
	///////////////////////////////////////////////////////////////////////////////////////////////////
	//hipGetLastError();
	// Reserve memory for return data "mapQ_ho" and "pos_ho" on the host.
	seqs_mapQ_ho = (uint8_t *) malloc(sizeof(uint8_t)*n_seqs_max);
	if (seqs_mapQ_ho == NULL) report_cuda_error_CPU("[samse_core] Error reserving memory for \"seqs_mapQ_ho\".\n");
	seqs_pos_ho = (bwtint_t *) malloc(sizeof(bwtint_t)*n_seqs_max);
	if (seqs_pos_ho == NULL) report_cuda_error_CPU("[samse_core] Error reserving memory for \"seqs_pos_ho\".\n");

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Reserve memory for output data variables "seqs_mapQ_de" and "seqs_pos_de" on device.
	///////////////////////////////////////////////////////////////////////////////////////////////////
	hipMalloc(&seqs_mapQ_de,sizeof(uint8_t)*n_seqs_max);
	report_cuda_error_GPU("[samse_core] Error reserving memory for \"seqs_mapQ_de\".\n");
	hipMalloc(&seqs_pos_de,sizeof(bwtint_t)*n_seqs_max);
	report_cuda_error_GPU("[samse_core] Error reserving memory for \"seqs_pos_de\".\n");

	size_t mem_available, total_mem;

	hipMemGetInfo(&mem_available, &total_mem);
	report_cuda_error_GPU("[samse_core] Error_2 on hipMemGetInfo");

#if DEBUG_LEVEL > 0
	fprintf(stderr,"[samse_debug] sequence loaded loaded mem left: %d MB\n", (int)(mem_available>>20));
#endif

}

void free_bwa_cal_pac_pos_cuda1(
    unsigned int *global_bwt,
    unsigned int *global_rbwt,
    bwtint_t *bwt_sa_de,
    bwtint_t *rbwt_sa_de,
    int *g_log_n_de)
{

    ////////////////////////////////////////////////////////////
    // Clean up data.
    ////////////////////////////////////////////////////////////
    // Erase BWT on GPU device.
    free_bwts_from_cuda_memory(global_bwt,global_rbwt);

    // Delete memory used.
    hipFree(bwt_sa_de);
    hipFree(rbwt_sa_de);
    hipFree(g_log_n_de);

    // Unbind texture to reads.
    //hipUnbindTexture(sa_tex);

    // Unbind texture to "g_log_n_tex".
    hipUnbindTexture(g_log_n_tex);

    // Unbind textures to BWT and RBWT.
    hipUnbindTexture(bwt_sa_tex);
    hipUnbindTexture(rbwt_sa_tex);

}

void free_bwa_cal_pac_pos_cuda2()
{
	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Clean up data.
	///////////////////////////////////////////////////////////////////////////////////////////////////
	free(seqs_maxdiff_mapQ_ho);
	hipFree(seqs_maxdiff_mapQ_de);
	free(seqs_sa_ho);
	hipFree(seqs_sa_de);
	free(seqs_pos_ho);
	hipFree(seqs_pos_de);
	free(seqs_mapQ_ho);
	hipFree(seqs_mapQ_de);
}



// This function is meant to be a GPU implementation of bwa_cal_pac_pos(). Currently,
// only the forward strand is being tested for bwt_sa(). After that, test the reverse
// strand. Lastly, make GPU implementations of bwa_cal_maxdiff() and bwa_approx_mapQ().
void launch_bwa_cal_pac_pos_cuda(
	const char *prefix,
	int n_seqs,
	bwa_seq_t *seqs,
	int max_mm,
	float fnr,
	int device)
{

	//fprintf(stderr, "bwt->n_sa: %u %i\n",bwt->n_sa, int(sizeof(bwt->n_sa)));
	//fprintf(stderr, "bwt->sa_intv: %u %i\n",bwt->sa_intv, int(sizeof(bwt->sa_intv)));
	//fprintf(stderr, "rbwt->sa_intv: %u %i\n",rbwt->sa_intv, int(sizeof(rbwt->sa_intv)));

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Declare and initiate variables.
	///////////////////////////////////////////////////////////////////////////////////////////////////

	hipDeviceProp_t prop;
	int n_block;
	int n_seq_per_block;
	int block_mod;

	// Obtain information on CUDA devices.
	hipGetDeviceProperties(&prop, device);
	report_cuda_error_GPU("[samse_core] Error on \"hipGetDeviceProperties\".");

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Allocate memory and copy reads in "seqs" to "seqs_maxdiff_mapQ_ho" and "seqs_sa_ho".
	///////////////////////////////////////////////////////////////////////////////////////////////////
	for (int i = 0; i < n_seqs; i++)
	{
		seqs_maxdiff_mapQ_ho[i].len = seqs[i].len;
		//seqs_maxdiff_mapQ_ho[i].strand_type = ((seqs[i].strand<<2) | seqs[i].type);
		seqs_maxdiff_mapQ_ho[i].strand = seqs[i].strand;
		seqs_maxdiff_mapQ_ho[i].type = seqs[i].type;
		seqs_maxdiff_mapQ_ho[i].n_mm = seqs[i].n_mm;
		seqs_maxdiff_mapQ_ho[i].c1 = seqs[i].c1;
		seqs_maxdiff_mapQ_ho[i].c2 = seqs[i].c2;
		seqs_sa_ho[i] = seqs[i].sa;
	}


	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy input data in "seqs_maxdiff_mapQ_ho" and "seqs_sa_ho" to device, and bind texture of
	// "seqs_sa_de".
	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy data from host to device.
	hipMemcpy(seqs_maxdiff_mapQ_de,seqs_maxdiff_mapQ_ho,sizeof(bwa_maxdiff_mapQ_t)*n_seqs,hipMemcpyHostToDevice);
	report_cuda_error_GPU("Error copying to \"seqs_maxdiff_mapQ_de\".\n");

	// Copy data from host to device.
	hipMemcpy(seqs_sa_de,seqs_sa_ho,sizeof(bwtint_t)*n_seqs,hipMemcpyHostToDevice);
	report_cuda_error_GPU("Error copying to \"seqs_sa_de\".\n");

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Process bwa_cal_pac_pos_cuda()
	///////////////////////////////////////////////////////////////////////////////////////////////////
	// No. of blocks.
	n_block = 2048;
	// No of sequences per block.
	n_seq_per_block = n_seqs / n_block;
	// Extra sequences for the last block.
	block_mod = n_seqs - n_seq_per_block * n_block;

	//fprintf(stderr,"N_MP %i n_block %i n_seq_per_block %i block_mod %i\n", N_MP, n_block, n_seq_per_block, block_mod);
	//fprintf(stderr,"n_seqs %i\n", n_seqs);


	// Set block and grid sizes.
	dim3 dimBlock(BLOCK_SIZE2);
	dim3 dimGrid(n_block);

	// Execute bwt_sa function.
	cuda_bwa_cal_pac_pos_parallel2 <<<dimGrid, dimBlock>>>(
		seqs_mapQ_de,
		seqs_pos_de,
		seqs_maxdiff_mapQ_de,
		seqs_sa_de,
		n_seqs,
		n_block,
		n_seq_per_block,
		block_mod,
		max_mm,
		fnr);

	report_cuda_error_GPU("[samse_core] Error running \"cuda_bwa_cal_pac_pos()\".\n");
	hipDeviceSynchronize(); //wait until kernel has had a chance to report error
	hipDeviceSynchronize();
	report_cuda_error_GPU("[samse_core] Error synchronizing after \"cuda_bwa_cal_pac_pos()\".\n");


	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy data of output data variables in "seqs_mapQ_de" and "seqs_pos_de" to host.
	///////////////////////////////////////////////////////////////////////////////////////////////////

	// hipGetLastError();
	// Return data to host.
	hipMemcpy(seqs_mapQ_ho, seqs_mapQ_de, sizeof(uint8_t)*n_seqs, hipMemcpyDeviceToHost);
	report_cuda_error_GPU("[samse_core] Error copying to \"seqs_mapQ_ho\".\n");
	hipMemcpy(seqs_pos_ho, seqs_pos_de, sizeof(bwtint_t)*n_seqs, hipMemcpyDeviceToHost);
	report_cuda_error_GPU("[samse_core] Error copying to \"seqs_pos_ho\".\n");

	///////////////////////////////////////////////////////////////////////////////////////////////////
	// Save output data variables to "seqs".
	///////////////////////////////////////////////////////////////////////////////////////////////////
	for (int i = 0; i < n_seqs; i++)
	{
	 	seqs[i].mapQ = seqs_mapQ_ho[i];
		seqs[i].seQ = seqs_mapQ_ho[i];
		seqs[i].pos = seqs_pos_ho[i];
	}
}

//CUDA DEVICE CODE STARTING FROM THIS LINE
// This function does not work because the pointer b->bwt is not set.
__device__ uint32_t _bwt_bwt(const bwt_t *b, bwtint_t k)
{
	return ((b)->bwt[(k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL)/16]);
	//return ((b)->bwt[(k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL >> 4)]);
}

__device__ uint32_t _bwt_bwt2(bwtint_t k)
{
	//int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL) / 16;
	int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL >> 4);
	uint4 four_integers = tex1Dfetch(bwt_occ_array,pos>>2);
	uint32_t one_integer;

	switch (pos & 0x3)
	{
		case 0: one_integer = four_integers.x; break;
		case 1: one_integer = four_integers.y; break;
		case 2: one_integer = four_integers.z; break;
		case 3: one_integer = four_integers.w; break;
	}

	return one_integer;
}



__device__ uint32_t _bwt_bwt3(bwtint_t k, texture<uint4, 1, hipReadModeElementType> *b)
{
	//int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL) / 16;
	int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL >> 4);
	uint4 four_integers = tex1Dfetch(*b,pos>>2);
	uint32_t one_integer;

	switch (pos & 0x3)
	{
		case 0: one_integer = four_integers.x; break;
		case 1: one_integer = four_integers.y; break;
		case 2: one_integer = four_integers.z; break;
		case 3: one_integer = four_integers.w; break;
	}

	return one_integer;
}



__device__ uint32_t _rbwt_bwt2(bwtint_t k)
{
    //int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL) / 16;
    int pos = (k)/OCC_INTERVAL*12 + 4 + ((k)%OCC_INTERVAL >> 4);
    uint4 four_integers = tex1Dfetch(rbwt_occ_array,pos>>2);
    uint32_t one_integer;

    switch (pos & 0x3)
    {
        case 0: one_integer = four_integers.x; break;
        case 1: one_integer = four_integers.y; break;
        case 2: one_integer = four_integers.z; break;
        case 3: one_integer = four_integers.w; break;
    }

    return one_integer;
}



// This function does not work because the pointer b->bwt is not set.
__device__ ubyte_t _bwt_B0(const bwt_t *b, bwtint_t k)
{
	uint32_t tmp = _bwt_bwt(b,k)>>((~(k)&0xf)<<1)&3;
	ubyte_t c = ubyte_t(tmp);
	return c;
}



__device__ ubyte_t _bwt_B02(bwtint_t k)
{
	uint32_t tmp = _bwt_bwt2(k)>>((~(k)&0xf)<<1)&3;
	ubyte_t c = ubyte_t(tmp);
	return c;
}



__device__ ubyte_t _rbwt_B02(bwtint_t k)
{
    uint32_t tmp = _rbwt_bwt2(k)>>((~(k)&0xf)<<1)&3;
    ubyte_t c = ubyte_t(tmp);
    return c;
}



__device__ ubyte_t _bwt_B03(bwtint_t k, texture<uint4, 1, hipReadModeElementType> *b)
{
	//uint32_t tmp = _bwt_bwt3(k,b)>>((~(k)&0xf)<<1)&3;
	//ubyte_t c = ubyte_t(tmp);
    //return c;
    return ubyte_t(_bwt_bwt3(k,b)>>((~(k)&0xf)<<1)&3);
}



__device__ uint32_t* _bwt_occ_intv(const bwt_t *b, bwtint_t k)
{
	return ((b)->bwt + (k)/OCC_INTERVAL*12);
}



__device__
int cuda_bwa_cal_maxdiff(int l, double err, double thres)
{
    double elambda = exp(-l * err);
    double sum, y = 1.0;
    int k, x = 1;
    for (k = 1, sum = elambda; k < 1000; ++k) {
        y *= l * err;
        x *= k;
        sum += elambda * y / x;
        if (1.0 - sum < thres) return k;
    }
    return 2;
}



__device__
int cuda_bwa_approx_mapQ(const bwa_maxdiff_mapQ_t *p, int mm)
{
    int n, g_log;
    if (p->c1 == 0) return 23;
    if (p->c1 > 1) return 0;
    if (p->n_mm == mm) return 25;
    if (p->c2 == 0) return 37;
    n = (p->c2 >= 255)? 255 : p->c2;
    g_log = tex1Dfetch(g_log_n_tex,n);

    return (23 < g_log)? 0 : 23 - g_log;
}



__device__
void update_indices(
    int *n_sa_processed,
    int *n_sa_remaining,
    int *n_sa_in_buf,
    int *n_sa_buf_empty)
{
    (*n_sa_processed)++;
    (*n_sa_remaining)--;
    (*n_sa_in_buf)--;
    (*n_sa_buf_empty)++;
}



__device__
void update_indices_in_parallel(
    int *n_sa_processed,
    int *n_sa_remaining,
    int *n_sa_in_buf,
    int *n_sa_buf_empty)
{
    atomicAdd(*&n_sa_processed,1);
    atomicSub(*&n_sa_remaining,1);
    atomicSub(*&n_sa_in_buf,1);
    atomicAdd(*&n_sa_buf_empty,1);
}



__device__
void fetch_read_new_in_parallel(
    bwa_maxdiff_mapQ_t *maxdiff_mapQ_buf,
    int16_t *sa_origin,
    const bwa_maxdiff_mapQ_t *seqs_maxdiff_mapQ_de,
    const int offset,
    int *n_sa_in_buf,
    int *n_sa_buf_empty,
    int *n_sa_processed,
    int *n_sa_remaining,
    int *sa_next_no,
    const int n_sa_total,
    const char strand)
{
    while (*sa_next_no < n_sa_total)
    {
        int read_no_new = atomicAdd(*&sa_next_no,1);

        if (read_no_new < n_sa_total)
        {
            // Get new read from global memory.
            *maxdiff_mapQ_buf = seqs_maxdiff_mapQ_de[offset+read_no_new];
            //sa_buf_arr[tid] = seqs_sa_de[offset+read_no_new];
            // Check whether read can be used.
            if ((*maxdiff_mapQ_buf).strand == strand && ((*maxdiff_mapQ_buf).type == BWA_TYPE_UNIQUE ||
                (*maxdiff_mapQ_buf).type == BWA_TYPE_REPEAT))
            {
                *sa_origin = read_no_new;
                //sa_return[tid] = 0;
                atomicAdd(*&n_sa_in_buf,1);
                atomicSub(*&n_sa_buf_empty,1);
                break;
            }
            else
            {
                atomicAdd(*&n_sa_processed,1);
                atomicSub(*&n_sa_remaining,1);
                // Show that read is not being used.
            }
        }
    }
}



__device__
void sort_reads(
    bwtint_t *sa_buf_arr,
    bwa_maxdiff_mapQ_t *maxdiff_mapQ_buf_arr,
    int16_t *sa_origin,
    bwtint_t *sa_return,
    const int *n_sa_in_buf,
    int *n_sa_in_buf_prev)
{
    int sa_empty_no = *n_sa_in_buf_prev;
    *n_sa_in_buf_prev = *n_sa_in_buf;

    for (int j = 0; j < sa_empty_no; j++)
    {
        if (sa_origin[j] == -1)
        {
            for (int k = sa_empty_no-1; k > j; k--)
            {
                sa_empty_no--;
                if (sa_origin[k] != -1)
                {
                    sa_buf_arr[j] = sa_buf_arr[k];
                    maxdiff_mapQ_buf_arr[j] = maxdiff_mapQ_buf_arr[k];
                    sa_origin[j] = sa_origin[k];
                    sa_return[j] = sa_return[k];
                    sa_origin[k] = -1;
                    break;
                }
            }
        }
    }
}

// This function can process a maximum of 2**15 reads per block.
// bwt_sa() with texture reads (alignment 1).
// BWT and RBWT are separated by order (run in succession).
__global__
void cuda_bwa_cal_pac_pos_parallel2(
    uint8_t *seqs_mapQ_de,
    bwtint_t *seqs_pos_de,
    const bwa_maxdiff_mapQ_t *seqs_maxdiff_mapQ_de,
    const bwtint_t *seqs_sa_de,
    int n_seqs,
    int n_block,
    int n_seq_per_block,
    int block_mod,
    int max_mm,
    float fnr)
{
    // Declare and initialize variables.
    // Thread ID and offset.
    const int tid = threadIdx.x;
    const int offset = blockIdx.x < block_mod ? (n_seq_per_block+1)*blockIdx.x : (n_seq_per_block+1)*block_mod + n_seq_per_block*(blockIdx.x-block_mod);
    const int n_sa_total = n_seq_per_block + (blockIdx.x < block_mod ? 1 : 0);

    int bwt_sa_intv = bwt_cuda.sa_intv;
    int rbwt_sa_intv = rbwt_cuda.sa_intv;

    __shared__ int n_sa_processed;
    __shared__ int n_sa_remaining;
    __shared__ int n_sa_in_buf;
    __shared__ int n_sa_in_buf_prev;
    __shared__ int n_sa_buf_empty;
    __shared__ int sa_next_no;

    __shared__ bwtint_t sa_buf_arr[BLOCK_SIZE2];    // Array of "sa".
    __shared__ bwa_maxdiff_mapQ_t maxdiff_mapQ_buf_arr[BLOCK_SIZE2];    // Array of "maxdiff" elements.
    __shared__ int16_t sa_origin[BLOCK_SIZE2];  // Index of reads.
    __shared__ bwtint_t sa_return[BLOCK_SIZE2]; // Return value.

    // "n_sa_total" is the total number of reads of the block, "n_sa_processed" is the number of finished
    // reads: "n_total = n_sa_processed + n_sa_remaining". "n_sa_in_buf" (<= BLOCK_SIZE2) is the number of
    // reads in process in the buffer, and "n_sa_buf_empty" is the number of empty elements in the buffer:
    // BUFFER_SIZE2 = n_sa_in_buf + n_sa_buf_empty". "sa_next_no" (< "n_total") is the number of the read
    // to fetch next from global or texture memory.

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Run BWT.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////

    n_sa_processed = 0;
    n_sa_remaining = n_sa_total;
    n_sa_in_buf = min(n_sa_total,BLOCK_SIZE2);
    n_sa_in_buf_prev = n_sa_in_buf;
    n_sa_buf_empty = BLOCK_SIZE2 - n_sa_in_buf;
    sa_next_no = n_sa_in_buf;

    __syncthreads();

    // Fill arrays with initial values. (Do this first to reduce latency as reading from global
    // memory is time-consuming).
    if (tid < n_sa_in_buf)
    {
        maxdiff_mapQ_buf_arr[tid] = seqs_maxdiff_mapQ_de[offset+tid];
        sa_buf_arr[tid] = seqs_sa_de[offset+tid];
    }

    // Set the position in the position array and state which threads are not in use (-1).
    sa_origin[tid] = tid < n_sa_in_buf ? tid : -1;

    // Initialize the return values
    sa_return[tid] = 0;

    // Get new reads on the right strand.
    if (tid < n_sa_in_buf &&
        !(maxdiff_mapQ_buf_arr[tid].strand && (maxdiff_mapQ_buf_arr[tid].type == BWA_TYPE_UNIQUE ||
        maxdiff_mapQ_buf_arr[tid].type == BWA_TYPE_REPEAT)))
    {
        update_indices_in_parallel(&n_sa_processed,&n_sa_remaining,&n_sa_in_buf,&n_sa_buf_empty);
        sa_origin[tid] = -1;

        fetch_read_new_in_parallel(
            &maxdiff_mapQ_buf_arr[tid],
            &sa_origin[tid],
            seqs_maxdiff_mapQ_de,
            offset,
            &n_sa_in_buf,
            &n_sa_buf_empty,
            &n_sa_processed,
            &n_sa_remaining,
            &sa_next_no,
            n_sa_total,
            1);

        if (sa_origin[tid] != -1)
        {
            sa_buf_arr[tid] = seqs_sa_de[offset+sa_origin[tid]];
                    //tex1Dfetch(sa_tex,offset+sa_origin[tid]);
            sa_return[tid] = 0;
        }
    }

    // Get rid of reads that are on the wrong strand, fetch new ones.
    __syncthreads();

    if (n_sa_in_buf < BLOCK_SIZE2 && tid == 0)
    {
        sort_reads(
            &sa_buf_arr[0],
            &maxdiff_mapQ_buf_arr[0],
            &sa_origin[0],
            &sa_return[0],
            &n_sa_in_buf,
            &n_sa_in_buf_prev);
    }

    __syncthreads();

    // Start bwt_sa() in a loop until all reads have been processed.
    while (true)
    {
        // Return finished reads, fetch new reads if possible. Run in parallel, not sequentially.
        if //(sa_origin[tid] != -1)
           (tid < n_sa_in_buf)
        {
            char continuation = 1;
            if (sa_buf_arr[tid] % bwt_sa_intv == 0) {continuation = 0;}
            else if (sa_buf_arr[tid] == bwt_cuda.primary)
            {
                sa_return[tid]++;
                sa_buf_arr[tid] = 0;
                continuation = 0;
            }

            if (!continuation)
            {
                int max_diff = cuda_bwa_cal_maxdiff(maxdiff_mapQ_buf_arr[tid].len,BWA_AVG_ERR,fnr);
                uint8_t mapQ = cuda_bwa_approx_mapQ(&maxdiff_mapQ_buf_arr[tid],max_diff);

                // Return read that is finished.
                seqs_pos_de[offset+sa_origin[tid]] = sa_return[tid] + tex1Dfetch(bwt_sa_tex,sa_buf_arr[tid]/bwt_sa_intv);
                // Return "mapQ".
                seqs_mapQ_de[offset+sa_origin[tid]] = mapQ;
                sa_origin[tid] = -1;

                // Update indices.
                update_indices_in_parallel(&n_sa_processed,&n_sa_remaining,&n_sa_in_buf,&n_sa_buf_empty);

                // Get new read.
                fetch_read_new_in_parallel(
                    &maxdiff_mapQ_buf_arr[tid],
                    &sa_origin[tid],
                    seqs_maxdiff_mapQ_de,
                    offset,
                    &n_sa_in_buf,
                    &n_sa_buf_empty,
                    &n_sa_processed,
                    &n_sa_remaining,
                    &sa_next_no,
                    n_sa_total,
                    1);

                if (sa_origin[tid] != -1)
                {
                    sa_buf_arr[tid] = seqs_sa_de[offset+sa_origin[tid]];
                            //tex1Dfetch(sa_tex,offset+sa_origin[tid]);
                    sa_return[tid] = 0;
                }
            }
        }

        __syncthreads();

        if (n_sa_remaining <= 0) break;

        // This section puts reads in the buffer first to allow full warps to be run.
        if (n_sa_in_buf < BLOCK_SIZE2)
        {
            if (tid == 0)
            {
                sort_reads(
                    &sa_buf_arr[0],
                    &maxdiff_mapQ_buf_arr[0],
                    &sa_origin[0],
                    &sa_return[0],
                    &n_sa_in_buf,
                    &n_sa_in_buf_prev);
            }

            __syncthreads();
        }

        sa_return[tid]++;

        if (tid < n_sa_in_buf)
        {
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // Start bwt_sa (bwtint_t bwt_sa(const bwt_t *bwt, bwtint_t k))
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////

            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // Start #define bwt_invPsi(bwt, k)
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // First conditional expression.
            // Moved to the section above where "else if (sa_arr[k] == bwt_cuda.primary)".

            // Second conditional expression.
            bwtint_t invPsi1 = sa_buf_arr[tid] < bwt_cuda.primary ? sa_buf_arr[tid] : sa_buf_arr[tid]-1;
            ubyte_t invPsi2 = _bwt_B02(invPsi1);
            invPsi1 = bwt_cuda_occ(sa_buf_arr[tid],invPsi2);
            sa_buf_arr[tid] = bwt_cuda.L2[invPsi2]+invPsi1;
        }
    }

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Run RBWT.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////
    //seqs_pos_de[offset+sa_origin[tid]] = bwt_cuda.seq_len - (maxdiff_mapQ_buf_arr[tid].len +
    //    sa_return[tid] + tex1Dfetch(rbwt_sa_tex,sa_buf_arr[tid]/rbwt_sa_intv));
    __syncthreads();

    n_sa_processed = 0;
    n_sa_remaining = n_sa_total;
    n_sa_in_buf = min(n_sa_total,BLOCK_SIZE2);
    n_sa_in_buf_prev = n_sa_in_buf;
    n_sa_buf_empty = BLOCK_SIZE2 - n_sa_in_buf;
    sa_next_no = n_sa_in_buf;

    __syncthreads();

    // Fill arrays with initial values. (Do this first to reduce latency as reading from global
    // memory is time-consuming).
    if (tid < n_sa_in_buf)
    {
        maxdiff_mapQ_buf_arr[tid] = seqs_maxdiff_mapQ_de[offset+tid];
        sa_buf_arr[tid] = seqs_sa_de[offset+tid];
    }

    // Set the position in the return array.
    sa_origin[tid] = tid < n_sa_in_buf ? tid : -1;

    // Initialize the return values.
    sa_return[tid] = 0;

    // Get new reads on the right strand.
    if (tid < n_sa_in_buf &&
        !(!maxdiff_mapQ_buf_arr[tid].strand && (maxdiff_mapQ_buf_arr[tid].type == BWA_TYPE_UNIQUE ||
        maxdiff_mapQ_buf_arr[tid].type == BWA_TYPE_REPEAT)))
    {
        update_indices_in_parallel(&n_sa_processed,&n_sa_remaining,&n_sa_in_buf,&n_sa_buf_empty);
        sa_origin[tid] = -1;

        fetch_read_new_in_parallel(
            &maxdiff_mapQ_buf_arr[tid],
            &sa_origin[tid],
            seqs_maxdiff_mapQ_de,
            offset,
            &n_sa_in_buf,
            &n_sa_buf_empty,
            &n_sa_processed,
            &n_sa_remaining,
            &sa_next_no,
            n_sa_total,
            0);

        if (sa_origin[tid] != -1)
        {
            sa_buf_arr[tid] = seqs_sa_de[offset+sa_origin[tid]];
            sa_return[tid] = 0;
        }
    }

    // Sort reads.
    __syncthreads();

    if (tid == 0)
    {
        sort_reads(
            &sa_buf_arr[0],
            &maxdiff_mapQ_buf_arr[0],
            &sa_origin[0],
            &sa_return[0],
            &n_sa_in_buf,
            &n_sa_in_buf_prev);
    }

    __syncthreads();

    // Start bwt_sa() in a loop until all reads have been processed.
    while (true)
    {
        // Return finished reads, fetch new reads if possible. Run in parallel, not sequentially.
        if //(sa_origin[tid] != -1)
           (tid < n_sa_in_buf)
        {
            char continuation = 1;
            if (sa_buf_arr[tid] % rbwt_sa_intv == 0) {continuation = 0;}
            else if (sa_buf_arr[tid] == rbwt_cuda.primary)
            {
                sa_return[tid]++;
                sa_buf_arr[tid] = 0;
                continuation = 0;
            }

            if (!continuation)
            {
                int max_diff = cuda_bwa_cal_maxdiff(maxdiff_mapQ_buf_arr[tid].len,BWA_AVG_ERR,fnr);
                uint8_t mapQ = cuda_bwa_approx_mapQ(&maxdiff_mapQ_buf_arr[tid],max_diff);

                // Return read that is finished.
                //seqs_pos_de[offset+sa_origin[tid]] = sa_return[tid] + tex1Dfetch(bwt_sa_tex,sa_buf_arr[tid]/bwt_sa_intv);
                seqs_pos_de[offset+sa_origin[tid]] = bwt_cuda.seq_len - (maxdiff_mapQ_buf_arr[tid].len +
                    sa_return[tid] + tex1Dfetch(rbwt_sa_tex,sa_buf_arr[tid]/rbwt_sa_intv));
                // Return "mapQ".
                seqs_mapQ_de[offset+sa_origin[tid]] = mapQ;
                sa_origin[tid] = -1;

                // Update indices.
                update_indices_in_parallel(&n_sa_processed,&n_sa_remaining,&n_sa_in_buf,&n_sa_buf_empty);

                // Get new read.
                fetch_read_new_in_parallel(
                    &maxdiff_mapQ_buf_arr[tid],
                    &sa_origin[tid],
                    seqs_maxdiff_mapQ_de,
                    offset,
                    &n_sa_in_buf,
                    &n_sa_buf_empty,
                    &n_sa_processed,
                    &n_sa_remaining,
                    &sa_next_no,
                    n_sa_total,
                    0);

                if (sa_origin[tid] != -1)
                {
                    sa_buf_arr[tid] = seqs_sa_de[offset+sa_origin[tid]];
                    sa_return[tid] = 0;
                }
            }
        }

        __syncthreads();

        if (n_sa_remaining <= 0) break;

        // This section puts reads in the buffer first to allow full warps to be run.
        if (n_sa_in_buf < BLOCK_SIZE2)
        {
            if (tid == 0)
            {
                sort_reads(
                    &sa_buf_arr[0],
                    &maxdiff_mapQ_buf_arr[0],
                    &sa_origin[0],
                    &sa_return[0],
                    &n_sa_in_buf,
                    &n_sa_in_buf_prev);
            }

            __syncthreads();
        }

        sa_return[tid]++;

        if (tid < n_sa_in_buf)
        {
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // Start bwt_sa (bwtint_t bwt_sa(const bwt_t *bwt, bwtint_t k))
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////

            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // Start #define bwt_invPsi(bwt, k)
            ///////////////////////////////////////////////////////////////////////////////////////////////////////////
            // First conditional expression.
            // Moved to the section above where "else if (sa_arr[k] == bwt_cuda.primary)".

            // Second conditional expression.
            bwtint_t invPsi1 = sa_buf_arr[tid] < rbwt_cuda.primary ? sa_buf_arr[tid] : sa_buf_arr[tid]-1;
            ubyte_t invPsi2 = _rbwt_B02(invPsi1);
            invPsi1 = rbwt_cuda_occ(sa_buf_arr[tid],invPsi2);
            sa_buf_arr[tid] = rbwt_cuda.L2[invPsi2]+invPsi1;
        }
    }
}

#endif
///////////////////////////////////////////////////////////////
// End CUDA samse_core
///////////////////////////////////////////////////////////////

//END CUDA DEVICE CODE


